#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include "mygpu.h"
#include "cuda_util.h"
#include "../geometry/geometry.h"
#include "../util/query_context.h"
#include "../tracing/partitioner.h"
#include "../tracing/workbench.h"



//__device__
//inline void lookup(workbench *bench, uint pid, uint curnode){
//
//	Point *p = bench->points+pid;
//
//	bool top = (p->y>bench->schema[curnode].mid_y-bench->config->y_buffer);
//	bool bottom = (p->y<=bench->schema[curnode].mid_y+bench->config->y_buffer);
//	bool left = (p->x<=bench->schema[curnode].mid_x+bench->config->x_buffer);
//	bool right = (p->x>bench->schema[curnode].mid_x-bench->config->x_buffer);
//	uint need_check = (bottom&&left)*1+(bottom&&right)*2+(top&&left)*4+(top&&right)*8;
//	for(int i=0;i<4;i++){
//		if((need_check>>i)&1){
//			if((bench->schema[curnode].children[i]&1)){
//				uint gid = bench->schema[curnode].children[i]>>1;
//				assert(gid<bench->num_grids);
//				uint offset = 0;
//				while(offset<bench->grids[gid*(bench->config->grid_capacity+1)]){
//					uint cu_index = atomicAdd(&bench->unit_lookup_counter, 1);
//					bench->unit_lookup[cu_index].pid = pid;
//					bench->unit_lookup[cu_index].gid = gid;
//					bench->unit_lookup[cu_index].offset = offset;
//					//printf("%d\t%d\t%d\n",pid,gid,offset);
//					offset += bench->config->zone_capacity;
//				}
//			}else{
//				lookup(bench, pid, bench->schema[curnode].children[i]>>1);
//			}
//		}
//	}
//}
//
//// with recursive call
//__global__
//void lookup_recursive_cuda(workbench *bench){
//	int pid = blockIdx.x*blockDim.x+threadIdx.x;
//	if(pid>=bench->config->num_objects){
//		return;
//	}
//	lookup(bench,pid,0);
//	return;
//}


//  partition with cuda
__global__
void partition_cuda(workbench *bench){
	int pid = blockIdx.x*blockDim.x+threadIdx.x;
	if(pid>=bench->config->num_objects){
		return;
	}

	// search the tree to get in which grid
	uint curnode = 0;
	uint gid = 0;

	Point *p = bench->points+pid;
	while(true){
		int loc = (p->y>bench->schema[curnode].mid_y)*2
								+(p->x>bench->schema[curnode].mid_x);
		uint child_offset = bench->schema[curnode].children[loc];
		// is leaf
		if(bench->schema[child_offset].isleaf){
			gid = bench->schema[child_offset].node_id;
			break;
		}
		curnode = child_offset;
	}
	uint *cur_grid = bench->grids+(bench->config->grid_capacity+1)*gid;
	bench->grid_assignment[pid] = gid;

	// insert current pid to proper memory space of the target gid
	// todo: consider the situation that grid buffer is too small
	uint cur_loc = atomicAdd(cur_grid,1);
	if(cur_loc<bench->config->grid_capacity){
		*(cur_grid+1+cur_loc) = pid;
	}else{
		atomicSub(cur_grid,1);
	}
}

__global__
void cleargrids_cuda(workbench *bench){
	int gid = blockIdx.x*blockDim.x+threadIdx.x;
	if(gid>=bench->num_grids){
		return;
	}
	*(bench->grids+(bench->config->grid_capacity+1)*gid) = 0;
}

__global__
void reset_bench_cuda(workbench *bench){
	bench->grid_lookup_counter = 0;
	bench->unit_lookup_counter = 0;
	bench->reaches_counter = 0;
}


__global__
void initstack_cuda(workbench *bench){
	int pid = blockIdx.x*blockDim.x+threadIdx.x;
	if(pid>=bench->config->num_objects){
		return;
	}
	uint stack_index = atomicAdd(&bench->stack_index[0],1);
	assert(stack_index<bench->stack_capacity);
	bench->lookup_stack[0][stack_index*2] = pid;
	bench->lookup_stack[0][stack_index*2+1] = 0;
}

__global__
void lookup_cuda(workbench *bench, uint stack_id, uint stack_size){

	int sid = blockIdx.x*blockDim.x+threadIdx.x;
	if(sid>=stack_size){
		return;
	}

	uint pid = bench->lookup_stack[stack_id][sid*2];
	uint curnode = bench->lookup_stack[stack_id][sid*2+1];
	Point *p = bench->points+pid;
	//swap between 0 and 1
	uint next_stack_id = !stack_id;

	// could be possibly in multiple children with buffers enabled
	bool top = (p->y>bench->schema[curnode].mid_y-bench->config->y_buffer);
	bool bottom = (p->y<=bench->schema[curnode].mid_y+bench->config->y_buffer);
	bool left = (p->x<=bench->schema[curnode].mid_x+bench->config->x_buffer);
	bool right = (p->x>bench->schema[curnode].mid_x-bench->config->x_buffer);

	uint need_check = (bottom&&left)*1+(bottom&&right)*2+(top&&left)*4+(top&&right)*8;
	for(int i=0;i<4;i++){
		if((need_check>>i)&1){
			uint child_offset = bench->schema[curnode].children[i];
			if(bench->schema[child_offset].isleaf){
				uint gid = bench->schema[child_offset].node_id;
				assert(gid<bench->num_grids);
				uint offset = 0;
				while(offset<bench->grids[gid*(bench->config->grid_capacity+1)]){
					uint cu_index = atomicAdd(&bench->unit_lookup_counter, 1);
					assert(cu_index<bench->unit_lookup_capacity);
					bench->unit_lookup[cu_index].pid = pid;
					bench->unit_lookup[cu_index].gid = gid;
					bench->unit_lookup[cu_index].offset = offset;
					//printf("%d\t%d\t%d\n",pid,gid,offset);
					offset += bench->config->zone_capacity;
				}
			}else{
				uint stack_index = atomicAdd(&bench->stack_index[next_stack_id],1);
				assert(stack_index<bench->stack_capacity);
				bench->lookup_stack[next_stack_id][stack_index*2] = pid;
				bench->lookup_stack[next_stack_id][stack_index*2+1] = child_offset;
			}
		}
	}
	if(sid==0){
		bench->stack_index[stack_id] = 0;
	}
}


__global__
void reachability_cuda(workbench *bench){

	// the objects in which grid need be processed
	int pairid = blockIdx.x*blockDim.x+threadIdx.x;
	if(pairid>=bench->unit_lookup_counter){
		return;
	}

	double max_dist = bench->config->reach_distance;
	uint pid = bench->unit_lookup[pairid].pid;
	uint gid = bench->unit_lookup[pairid].gid;
	uint offset = bench->unit_lookup[pairid].offset;
	uint size = *(bench->grids+(bench->config->grid_capacity+1)*gid)-offset;

	if(size>bench->config->zone_capacity){
		size = bench->config->zone_capacity;
	}
	//printf("%d\t%d\t%d\t%d\n",pid,gid,offset,size);

	const uint *cur_pids = bench->grids+(bench->config->grid_capacity+1)*gid+1+offset;
	for(uint i=0;i<size;i++){
		if(pid!=cur_pids[i]){
			double dist = distance(bench->points[pid].x, bench->points[pid].y, bench->points[cur_pids[i]].x, bench->points[cur_pids[i]].y);
			if(dist<=max_dist){
				uint loc = atomicAdd(&bench->reaches_counter, 1);
				assert(loc<bench->reaches_capacity);
				bench->reaches[loc].pid1 = pid;
				bench->reaches[loc].pid2 = cur_pids[i];
			}
		}
	}
}

/*
 * in this phase, only update or append
 * */
__global__
void update_meetings_cuda(workbench *bench){

	int rid = blockIdx.x*blockDim.x+threadIdx.x;
	if(rid>=bench->reaches_counter){
		return;
	}
	uint pid1 = bench->reaches[rid].pid1;
	uint pid2 = bench->reaches[rid].pid2;
	uint bid = (pid1+pid2)%bench->config->num_meeting_buckets;
	meeting_unit *bucket = bench->meeting_buckets+bid*bench->meeting_bucket_capacity;
	bool updated = false;

	for(uint i=0;i<bench->meeting_buckets_counter_tmp[bid];i++){
		// a former meeting is encountered, update it
		if(bucket[i].pid1==pid1&&bucket[i].pid2==pid2){
			bucket[i].end = bench->cur_time;
			updated = true;
			break;
		}
	}

	// otherwise append it
	if(!updated){
		uint loc = atomicAdd(bench->meeting_buckets_counter+bid,1);
		assert(loc<bench->meeting_bucket_capacity);
		bucket[loc].pid1 = pid1;
		bucket[loc].pid2 = pid2;
		bucket[loc].start = bench->cur_time;
		bucket[loc].end = bench->cur_time;
	}
}

__global__
void compact_meetings_cuda(workbench *bench){
	int bid = blockIdx.x*blockDim.x+threadIdx.x;
	if(bid>=bench->config->num_meeting_buckets){
		return;
	}
	meeting_unit *bucket = bench->meeting_buckets+bid*bench->meeting_bucket_capacity;
	int front_idx = 0;
	int back_idx = bench->meeting_buckets_counter[bid]-1;
	uint meeting_idx = 0;
	int active_count = 0;
	for(;front_idx<=back_idx;front_idx++){
		// this meeting is over
		if(bucket[front_idx].end<bench->cur_time){
			// dump to valid list and copy one from the back end
			if(bucket[front_idx].end-bucket[front_idx].start>=bench->config->min_meet_time){
				meeting_idx = atomicAdd(&bench->meeting_counter,1);
				bench->meetings[meeting_idx] = bucket[front_idx];
			}
			for(;back_idx>front_idx;back_idx--){
				if(bucket[back_idx].end==bench->cur_time){
					break;
					// dump to valid list if needed or disregarded
				}else if(bucket[back_idx].end-bucket[back_idx].start>=bench->config->min_meet_time){
					meeting_idx = atomicAdd(&bench->meeting_counter,1);
					bench->meetings[meeting_idx] = bucket[back_idx];
				}
			}
			if(front_idx<back_idx){
				bucket[front_idx] = bucket[back_idx];
				active_count++;
				back_idx--;
			}
		}else{
			active_count++;
		}
	}
	bench->meeting_buckets_counter[bid] = active_count;
	bench->meeting_buckets_counter_tmp[bid] = active_count;
}

workbench *create_device_bench(workbench *bench, gpu_info *gpu){
	struct timeval start = get_cur_time();
	gpu->clear();
	// use h_bench as a container to copy in and out GPU
	workbench h_bench(bench);
	// space for the raw points data
	h_bench.points = (Point *)gpu->allocate(bench->config->num_objects*sizeof(Point));
	// space for the grid assignment information of each object
	h_bench.grid_assignment = (uint *)gpu->allocate(bench->config->num_objects*sizeof(uint));
	// space for the pids of all the grids
	h_bench.grids = (uint *)gpu->allocate(bench->num_grids*(bench->config->grid_capacity+1)*sizeof(uint));

	// space for the gid lookups
	h_bench.grid_lookup = (uint *)gpu->allocate(bench->grid_lookup_capacity*2*sizeof(uint));

	// space for the pid-zid pairs
	h_bench.unit_lookup = (checking_unit *)gpu->allocate(bench->unit_lookup_capacity*sizeof(checking_unit));
	// space for the QTtree schema
	h_bench.schema = (QTSchema *)gpu->allocate(bench->num_nodes*sizeof(QTSchema));
	// space for processing stack
	h_bench.lookup_stack[0] = (uint *)gpu->allocate(bench->stack_capacity*2*sizeof(uint));
	h_bench.lookup_stack[1] = (uint *)gpu->allocate(bench->stack_capacity*2*sizeof(uint));
	h_bench.reaches = (reach_unit *)gpu->allocate(bench->reaches_capacity*sizeof(reach_unit));

	h_bench.meeting_buckets = (meeting_unit *)gpu->allocate(bench->config->num_meeting_buckets*bench->meeting_bucket_capacity*sizeof(meeting_unit));
	h_bench.meeting_buckets_counter = (uint *)gpu->allocate(bench->config->num_meeting_buckets*sizeof(uint));
	h_bench.meeting_buckets_counter_tmp = (uint *)gpu->allocate(bench->config->num_meeting_buckets*sizeof(uint));
	h_bench.meetings = (meeting_unit *)gpu->allocate(bench->meeting_capacity*sizeof(meeting_unit));

	h_bench.config = (configuration *)gpu->allocate(sizeof(configuration));

	// space for the mapping of bench in GPU
	workbench *d_bench = (workbench *)gpu->allocate(sizeof(workbench));

	// the configuration and schema are fixed
	CUDA_SAFE_CALL(hipMemcpy(h_bench.schema, bench->schema, bench->num_nodes*sizeof(QTSchema), hipMemcpyHostToDevice));
	CUDA_SAFE_CALL(hipMemcpy(h_bench.config, bench->config, sizeof(configuration), hipMemcpyHostToDevice));
	CUDA_SAFE_CALL(hipMemcpy(d_bench, &h_bench, sizeof(workbench), hipMemcpyHostToDevice));

	logt("allocating space %d MB", start,gpu->size_allocated()/1024/1024);

	return d_bench;
}

/*
 *
 * check the reachability of objects in a list of partitions
 * ctx.data contains the list of
 *
 * */
void process_with_gpu(workbench *bench, workbench* d_bench, gpu_info *gpu){
	struct timeval start = get_cur_time();
	//gpu->print();
	assert(bench);
	assert(d_bench);
	assert(gpu);

	hipSetDevice(gpu->device_id);
	// as temporary host workbench
	workbench h_bench(bench);
	CUDA_SAFE_CALL(hipMemcpy(&h_bench, d_bench, sizeof(workbench), hipMemcpyDeviceToHost));
	h_bench.cur_time = bench->cur_time;
	CUDA_SAFE_CALL(hipMemcpy(d_bench, &h_bench, sizeof(workbench), hipMemcpyHostToDevice));

	CUDA_SAFE_CALL(hipMemcpy(h_bench.points, bench->points, bench->config->num_objects*sizeof(Point), hipMemcpyHostToDevice));
	logt("copying data", start);

	partition_cuda<<<bench->config->num_objects/1024+1,1024>>>(d_bench);
	check_execution();
	hipDeviceSynchronize();
	logt("partition data", start);

	initstack_cuda<<<bench->config->num_objects/1024+1,1024>>>(d_bench);
	check_execution();
	hipDeviceSynchronize();
	uint stack_id = 0;
	h_bench.stack_index[stack_id] = bench->config->num_objects;
	while(h_bench.stack_index[stack_id]>0){
		lookup_cuda<<<h_bench.stack_index[stack_id]/1024+1,1024>>>(d_bench,stack_id,h_bench.stack_index[stack_id]);
		check_execution();
		hipDeviceSynchronize();
		CUDA_SAFE_CALL(hipMemcpy(&h_bench, d_bench, sizeof(workbench), hipMemcpyDeviceToHost));
		stack_id = !stack_id;
	}
	logt("%d pid-grid pairs need is retrieved", start,h_bench.unit_lookup_counter);

	// compute the reachability of objects in each partitions
	reachability_cuda<<<h_bench.unit_lookup_counter/1024+1,1024>>>(d_bench);
	check_execution();
	hipDeviceSynchronize();
	CUDA_SAFE_CALL(hipMemcpy(&h_bench, d_bench, sizeof(workbench), hipMemcpyDeviceToHost));
	logt("%d reaches are found", start,h_bench.reaches_counter);

	update_meetings_cuda<<<h_bench.reaches_counter/1024+1,1024>>>(d_bench);
	check_execution();
	hipDeviceSynchronize();
	CUDA_SAFE_CALL(hipMemcpy(&h_bench, d_bench, sizeof(workbench), hipMemcpyDeviceToHost));
	logt("meeting buckets are updated", start);

	compact_meetings_cuda<<<bench->config->num_meeting_buckets/1024+1,1024>>>(d_bench);
	check_execution();
	hipDeviceSynchronize();
	CUDA_SAFE_CALL(hipMemcpy(&h_bench, d_bench, sizeof(workbench), hipMemcpyDeviceToHost));
	logt("meeting buckets are compacted %d meetings are found", start, h_bench.meeting_counter);

	// todo for test only, should not copy out so much stuff
	if(bench->config->analyze){
		CUDA_SAFE_CALL(hipMemcpy(bench->grids, h_bench.grids,
				bench->num_grids*(bench->config->grid_capacity+1)*sizeof(uint), hipMemcpyDeviceToHost));
		CUDA_SAFE_CALL(hipMemcpy(bench->meeting_buckets, h_bench.meeting_buckets,
				bench->config->num_meeting_buckets*bench->meeting_bucket_capacity*sizeof(meeting_unit), hipMemcpyDeviceToHost));
		CUDA_SAFE_CALL(hipMemcpy(bench->meeting_buckets_counter, h_bench.meeting_buckets_counter,
				bench->config->num_meeting_buckets*sizeof(uint), hipMemcpyDeviceToHost));
	}
	if(h_bench.meeting_counter>0){
		bench->meeting_counter = h_bench.meeting_counter;
		CUDA_SAFE_CALL(hipMemcpy(bench->meetings, h_bench.meetings,
				h_bench.meeting_counter*sizeof(meeting_unit), hipMemcpyDeviceToHost));
	}
	logt("copy out", start);
	// clean the device bench for next round of checking
	cleargrids_cuda<<<bench->num_grids/1024+1,1024>>>(d_bench);
	//clear_meeting_buckets_cuda<<<bench->config->num_meeting_buckets/1024+1,1024>>>(d_bench);
	reset_bench_cuda<<<1,1>>>(d_bench);
	logt("clean", start);
}
