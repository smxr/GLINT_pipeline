#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include "mygpu.h"
#include "cuda_util.h"
#include "../geometry/geometry.h"
#include "../util/query_context.h"

// return the distance of two segments

const static double degree_per_meter_latitude_cuda = 360.0/(40076.0*1000);

__device__
inline double degree_per_meter_longitude_cuda(double latitude){
	return 360.0/(sin((90-abs(latitude))*PI/180)*40076.0*1000.0);
}

__device__
inline double distance(const double x1, const double y1, const double x2, const double y2){
	double dx = x1-x2;
	double dy = y1-y2;
	dx = dx/degree_per_meter_longitude_cuda(y1);
	dy = dy/degree_per_meter_latitude_cuda;
	return sqrt(dx*dx+dy*dy);
}

__global__
void reachability_cuda(const double *points, const uint *gridcheck, uint *pids, const offset_size *os, uint *ret, size_t num_checkes, double max_dist){

	// the objects in which grid need be processed
	int pairid = blockIdx.x*blockDim.x+threadIdx.x;
	if(pairid>=num_checkes){
		return;
	}

	uint pid = gridcheck[pairid*2];
	uint gid = gridcheck[pairid*2+1];

	const uint *cur_pids = pids+os[gid].offset;
	double curx = *(points+pid*2);
	double cury = *(points+pid*2+1);
	//printf("%d %d %d\n",pid,gid,ret[pid]);
	for(uint i=0;i<os[gid].size;i++){
		if(pid!=cur_pids[i]){
			double dist = distance(curx, cury,*(points+cur_pids[i]*2),*(points+cur_pids[i]*2+1));
			//printf("%d %d %d\n",pid,gid,ret[pid]);
			ret[pid] += dist<=max_dist;
			ret[cur_pids[i]] += dist<=max_dist;
		}
	}
	if(ret[pid]>1000){
		printf("%d\t%d\n",pid,ret[pid]);
	}
}


query_context partition_with_gpu(Point *points, size_t num_objects, offset_size *os){

	query_context ctx;

	return ctx;
}

__global__ void mykernel(Point *p1, Point *p2, double *dist) {
	*dist = distance(p1->x,p1->y,p2->x,p2->y);
	printf("gpu %f\n",*dist);
  //*addr += 10;
  //atomicAdd(addr, 10);       // only available on devices with compute capability 6.x
}

int foo(Point *p1, Point *p2) {
	Point *d_p1,*d_p2;
	double *d_dist;
	hipMallocManaged(&d_p1, sizeof(Point));
	hipMallocManaged(&d_p2, sizeof(Point));
	hipMallocManaged(&d_dist, sizeof(double));

	CUDA_SAFE_CALL(hipMemcpy(d_p1, p1, sizeof(Point), hipMemcpyHostToDevice));
	CUDA_SAFE_CALL(hipMemcpy(d_p2, p2, sizeof(Point), hipMemcpyHostToDevice));

	mykernel<<<1,1>>>(d_p1, d_p2, d_dist);
	double dist = 0;
	CUDA_SAFE_CALL(hipMemcpy(&dist, d_dist, sizeof(double), hipMemcpyDeviceToHost));
	int ret = 0;
	return ret;
}


/*
 *
 * check the reachability of objects in a list of partitions
 * ctx.data contains the list of
 *
 * */
void process_with_gpu(query_context &ctx){
	struct timeval start = get_cur_time();

	vector<gpu_info *> gpus = get_gpus();
	gpu_info *gpu = gpus[0];

	pthread_mutex_lock(&gpu->lock);
	assert(gpu);
	hipSetDevice(gpu->device_id);

	Point *points = (Point *)ctx.target[0];
	uint *partitions = (uint *)ctx.target[1];
	offset_size *os = (offset_size *)ctx.target[2];
	uint *result = (uint *)ctx.target[3];
	uint *grid_checks = (uint *)ctx.target[4];

	size_t num_points = ctx.target_length[0];
	size_t num_grids = ctx.target_length[2];
	size_t num_checkes = ctx.target_length[4];

	// space for the raw points data
	Point *d_points = (Point *)gpu->get_data(0, sizeof(Point)*num_points);
	// space for the pids of all the grids
	uint *d_partitions = (uint *)gpu->get_data(1, num_points*sizeof(uint));
	// space for the offset and size information in GPU
	offset_size *d_os = (offset_size *)gpu->get_data(2, sizeof(offset_size)*num_grids);
	// space for the results in GPU
	uint *d_ret = (uint *)gpu->get_data(3, sizeof(uint)*num_points);

	uint *d_gridcheck = (uint *)gpu->get_data(4, 2*sizeof(uint)*num_checkes);
	logt("allocating space", start);

	CUDA_SAFE_CALL(hipMemcpy(d_points, points, num_points*sizeof(Point), hipMemcpyHostToDevice));
	CUDA_SAFE_CALL(hipMemcpy(d_partitions, partitions, num_points*sizeof(uint), hipMemcpyHostToDevice));
	CUDA_SAFE_CALL(hipMemcpy(d_os, os, num_grids*sizeof(offset_size), hipMemcpyHostToDevice));
	CUDA_SAFE_CALL(hipMemcpy(d_gridcheck, grid_checks, num_checkes*2*sizeof(uint), hipMemcpyHostToDevice));

	logt("copying data", start);
	// compute the reachability of objects in each partitions
	reachability_cuda<<<num_checkes/1024+1,1024>>>((double *)d_points, d_gridcheck, d_partitions, d_os, d_ret, num_checkes, ctx.config.reach_distance);

	check_execution();
	hipDeviceSynchronize();
	CUDA_SAFE_CALL(hipMemcpy(result, d_ret, num_points*sizeof(uint), hipMemcpyDeviceToHost));
	pthread_mutex_unlock(&gpu->lock);
	for(gpu_info *g:gpus){
		delete g;
	}
	ctx.found = 0;
	for(int i=0;i<num_points;i++){
		ctx.found += result[i];
	}
	logt("computing with GPU", start);
}

