#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include "mygpu.h"
#include "cuda_util.cuh"
#include "hilbert_curve.cuh"
#include "../geometry/geometry.h"
#include "../util/query_context.h"
#include "../tracing/partitioner.h"
#include "../tracing/workbench.h"
#include <thrust/host_vector.h>
#include <thrust/device_vector.h>
#include <thrust/generate.h>
#include <thrust/sort.h>
#include <thrust/copy.h>

/*
 *
 * some utility functions
 *
 * */

__device__
inline double height(box *b){
	return (b->high[1]-b->low[1])/degree_per_meter_latitude_cuda;
}

__device__
inline double distance(box *b,Point *p){

	double dx = max(abs(p->x-(b->low[0]+b->high[0])/2) - (b->high[0]-b->low[0])/2, 0.0);
	double dy = max(abs(p->y-(b->low[1]+b->high[1])/2) - (b->high[1]-b->low[1])/2, 0.0);
	dy = dy/degree_per_meter_latitude_cuda;
	dx = dx/degree_per_meter_longitude_cuda(p->y);

	return sqrt(dx * dx + dy * dy);
}

__device__
inline double contain(box *b, Point *p){
	return p->x>=b->low[0]&&
		   p->x<=b->high[0]&&
		   p->y>=b->low[1]&&
		   p->y<=b->high[1];
}

__device__
inline void print_box_point(box *b, Point *p){
	printf("POLYGON((%f %f, %f %f, %f %f, %f %f, %f %f))\nPOINT(%f %f)\n",
						b->low[0],b->low[1],
						b->high[0],b->low[1],
						b->high[0],b->high[1],
						b->low[0],b->high[1],
						b->low[0],b->low[1],
						p->x,p->y);
}

__device__
inline void print_box(box *b){
	printf("POLYGON((%f %f, %f %f, %f %f, %f %f, %f %f))\n",
						b->low[0],b->low[1],
						b->high[0],b->low[1],
						b->high[0],b->high[1],
						b->low[0],b->high[1],
						b->low[0],b->low[1]);
}

__device__
inline void mbr_update(box &mbr, Point *p){
    if(mbr.low[0]>p->x){
        mbr.low[0] = p->x;
    }
    if(mbr.high[0]<p->x){
        mbr.high[0] = p->x;
    }

    if(mbr.low[1]>p->y){
        mbr.low[1] = p->y;
    }
    if(mbr.high[1]<p->y){
        mbr.high[1] = p->y;
    }
}

__device__
inline void print_point(Point *p){
	printf("Point(%f %f)\n",p->x,p->y);
}

__device__
__uint128_t box_to_128(box *b){
    return ((__uint128_t)float_to_uint(b->low[0]) << 66) + ((__uint128_t)float_to_uint(b->low[1]) << 44) + ((__uint128_t)float_to_uint(b->high[0]) << 22) + ((__uint128_t)float_to_uint(b->high[1]));
}

/*
 *
 * kernel functions
 *
 * */

__global__
void cuda_cleargrids(workbench *bench){
	int gid = blockIdx.x*blockDim.x+threadIdx.x;
	if(gid>=bench->grids_stack_capacity){
		return;
	}
	bench->grid_counter[gid] = 0;
}

__global__
void cuda_reset_bench(workbench *bench){
	bench->grid_check_counter = 0;
	//bench->meeting_counter = 0;
	bench->num_active_meetings = 0;
	bench->num_taken_buckets = 0;
	bench->filter_list_index = 0;
	bench->split_list_index = 0;
	bench->merge_list_index = 0;
}

__global__
void cuda_clean_buckets(workbench *bench){
	size_t bid = blockIdx.x*blockDim.x+threadIdx.x;
	if(bid>=bench->config->num_meeting_buckets){
		return;
	}
	bench->meeting_buckets[bid].key = ULL_MAX;
}



//  partition with cuda
__global__
void cuda_partition(workbench *bench){
	int pid = blockIdx.x*blockDim.x+threadIdx.x;
	if(pid>=bench->config->num_objects){
		return;
	}

	// search the tree to get in which grid
	uint curnode = 0;
	uint gid = 0;

	Point *p = bench->points+pid;
	uint last_valid = 0;
	while(true){
		int loc = (p->y>bench->schema[curnode].mid_y)*2 + (p->x>bench->schema[curnode].mid_x);
		curnode = bench->schema[curnode].children[loc];

		// not near the right and top border
		if(p->x+bench->config->x_buffer<bench->schema[curnode].mbr.high[0]&&
		   p->y+bench->config->y_buffer<bench->schema[curnode].mbr.high[1]){
			last_valid = curnode;
		}

		// is leaf
		if(bench->schema[curnode].type==LEAF){
			gid = bench->schema[curnode].grid_id;
			break;
		}
	}

	// insert current pid to proper memory space of the target gid
	// todo: consider the situation that grid buffer is too small
	uint *cur_grid = bench->grids+bench->grid_capacity*gid;
	uint cur_loc = atomicAdd(bench->grid_counter+gid,1);
	if(cur_loc<bench->grid_capacity){
		*(cur_grid+cur_loc) = pid;
	}
	uint glid = atomicAdd(&bench->grid_check_counter,1);
	bench->grid_check[glid].pid = pid;
	bench->grid_check[glid].gid = gid;
	bench->grid_check[glid].offset = 0;
	bench->grid_check[glid].inside = true;

	if(last_valid!=curnode){
		uint stack_index = atomicAdd(&bench->filter_list_index,1);
		assert(stack_index<bench->filter_list_capacity);
		bench->filter_list[stack_index*2] = pid;
		bench->filter_list[stack_index*2+1] = last_valid;
	}

}


/*
 *
 * functions for filtering
 *
 * */

#define PER_STACK_SIZE 5

__global__
void cuda_pack_lookup(workbench *bench){
	int pid = blockIdx.x*blockDim.x+threadIdx.x;
	if(pid>=bench->config->num_objects){
		return;
	}

	uint idx = atomicAdd(&bench->filter_list_index,1);
	assert(idx<bench->filter_list_capacity);
	bench->filter_list[idx*2] = pid;
	bench->filter_list[idx*2+1] = 0;
}

__global__
void cuda_filtering(workbench *bench, int start_idx, int batch_size, bool include_contain){
	int cur_idx = blockIdx.x*blockDim.x+threadIdx.x;
	int idx = cur_idx + start_idx;
	if(cur_idx>=batch_size){
		return;
	}
	int pid = bench->filter_list[idx*2];
	int nodeid = bench->filter_list[idx*2+1];

	// get the block shared stack
	int block_stack_size = 1024*2*PER_STACK_SIZE;
	int stack_offset = blockIdx.x*block_stack_size;

	assert(stack_offset+block_stack_size<bench->tmp_space_capacity);

	int *cur_stack_idx = (int *)bench->tmp_space+stack_offset;
	int *cur_worker_idx = (int *)bench->tmp_space+stack_offset+1;
	uint *cur_stack = bench->tmp_space+stack_offset+2;

	*cur_stack_idx = 0;
	*cur_worker_idx = 0;
	__syncthreads();

	int stack_index = atomicAdd(cur_stack_idx, 1);
	cur_stack[2*stack_index] = pid;
	cur_stack[2*stack_index+1] = nodeid;

	//printf("%d:\tinit push %d\n",threadIdx.x,stack_index);
	__syncthreads();

	while(true){
		bool busy = false;
		stack_index = atomicSub(cur_stack_idx, 1)-1;
		//printf("%d:\tpop %d\n",threadIdx.x, stack_index);
		__syncthreads();
		if(stack_index<0){
			stack_index = atomicAdd(cur_stack_idx, 1);
			//printf("%d:\tinc %d\n",threadIdx.x, stack_index);
		}else{
			busy = true;
			atomicAdd(cur_worker_idx, 1);
		}
		__syncthreads();

		//printf("num workers: %d\n",*cur_worker_idx);
		if(*cur_worker_idx==0){
			break;
		}
		if(busy){

			uint pid = cur_stack[2*stack_index];
			uint curnode = cur_stack[2*stack_index+1];
			Point *p = bench->points+pid;
			//printf("process: %d %d %d\n",stack_index,pid,curnode);

			for(int i=0;i<4;i++){
				uint child_offset = bench->schema[curnode].children[i];
				double dist = distance(&bench->schema[child_offset].mbr, p);
				if(dist<=bench->config->reach_distance){
					if(bench->schema[child_offset].type==LEAF){
						uint gid = bench->schema[child_offset].grid_id;
						assert(gid<bench->grids_stack_capacity);
						if(include_contain&&contain(&bench->schema[child_offset].mbr,p)){
							uint *cur_grid = bench->grids+bench->grid_capacity*gid;
							uint cur_loc = atomicAdd(bench->grid_counter+gid,1);
							if(cur_loc<bench->grid_capacity){
								*(cur_grid+cur_loc) = pid;
							}
							uint glid = atomicAdd(&bench->grid_check_counter,1);
							assert(glid<bench->grid_check_capacity);
							bench->grid_check[glid].pid = pid;
							bench->grid_check[glid].gid = gid;
							bench->grid_check[glid].offset = 0;
							bench->grid_check[glid].inside = true;
						}else if(p->y<bench->schema[child_offset].mbr.low[1]||
						   (p->y<bench->schema[child_offset].mbr.high[1]
							&& p->x<bench->schema[child_offset].mbr.low[0])){
							uint glid = atomicAdd(&bench->grid_check_counter,1);
							assert(glid<bench->grid_check_capacity);
							bench->grid_check[glid].pid = pid;
							bench->grid_check[glid].gid = gid;
							bench->grid_check[glid].offset = 0;
							bench->grid_check[glid].inside = false;
						}
					}else{
						stack_index = atomicAdd(cur_stack_idx, 1);
						//printf("%d:\tnew push %d\n",threadIdx.x,stack_index);
						assert(stack_index<PER_STACK_SIZE*1024);
						cur_stack[2*stack_index] = pid;
						cur_stack[2*stack_index+1] = child_offset;
					}
				}
			}
			atomicSub(cur_worker_idx, 1);
		}
		__syncthreads();
	}
}


/*
 *
 * kernel functions for the refinement step
 *
 * */

__global__
void cuda_unroll(workbench *bench, uint inistial_size){
	int glid = blockIdx.x*blockDim.x+threadIdx.x;
	if(glid>=inistial_size){
		return;
	}

	uint grid_size = min(bench->grid_counter[bench->grid_check[glid].gid],bench->grid_capacity);
	// the first batch already inserted during the partition and lookup steps
	uint offset = bench->config->zone_capacity;
	while(offset<grid_size){
		uint cu_index = atomicAdd(&bench->grid_check_counter, 1);
		if(cu_index>=bench->grid_check_capacity){
			printf("%d %d %d\n",bench->grid_counter[bench->grid_check[glid].gid],cu_index,bench->grid_check_capacity);
		}
		//assert(cu_index<bench->grid_check_capacity);
		bench->grid_check[cu_index] = bench->grid_check[glid];
		bench->grid_check[cu_index].offset = offset;
		offset += bench->config->zone_capacity;
	}
}


__global__
void cuda_refinement(workbench *bench){

    // the objects in which grid need be processed
    int loc = threadIdx.y;
    int pairid = blockIdx.x*blockDim.x+threadIdx.x;
    if(pairid>=bench->grid_check_counter){
        return;
    }

    uint gid = bench->grid_check[pairid].gid;
    uint offset = bench->grid_check[pairid].offset;

    uint size = min(bench->grid_counter[gid],bench->grid_capacity)-offset;
    if(bench->config->unroll && size>bench->config->zone_capacity){
        size = bench->config->zone_capacity;
    }
    if(loc>=size){
        return;
    }
    uint pid = bench->grid_check[pairid].pid;
    uint target_pid = *(bench->grids+bench->grid_capacity*gid+offset+loc);
    if(!bench->grid_check[pairid].inside||pid<target_pid){
//        Point *p1 = &bench->points[pid];
//        Point *p2 = &bench->points[target_pid];
        Point *p1 = bench->points+pid;
        Point *p2 = bench->points+target_pid;
        double dist = distance(bench->points[pid].x, bench->points[pid].y, bench->points[target_pid].x, bench->points[target_pid].y);
        if(dist<=bench->config->reach_distance){
            uint pid1 = min(pid,target_pid);
            uint pid2 = max(target_pid,pid);
            size_t key = ((size_t)pid1+pid2)*(pid1+pid2+1)/2+pid2;
            size_t slot = key%bench->config->num_meeting_buckets;
            int ite = 0;
            while (ite++<5){
                unsigned long long prev = atomicCAS((unsigned long long *)&bench->meeting_buckets[slot].key, ULL_MAX, (unsigned long long)key);
                //printf("%ld\n",prev,ULL_MAX,bench->meeting_buckets[bench->current_bucket][slot].key);
                if(prev == key){
                    bench->meeting_buckets[slot].end = bench->cur_time;
                    //mbr_update(bench->meeting_buckets[slot].mbr, bench->points[pid]);                     //"Point::~Point"
                    mbr_update(bench->meeting_buckets[slot].mbr, p1);
                    mbr_update(bench->meeting_buckets[slot].mbr, p2);
                    break;
                }else if (prev == ULL_MAX){
                    bench->meeting_buckets[slot].key = key;
                    bench->meeting_buckets[slot].start = bench->cur_time;
                    bench->meeting_buckets[slot].end = bench->cur_time;
                    bench->meeting_buckets[slot].mbr.low[0] = 100000.0;
                    bench->meeting_buckets[slot].mbr.low[1] = 100000.0;
                    bench->meeting_buckets[slot].mbr.high[0] = -100000.0;
                    bench->meeting_buckets[slot].mbr.high[1] = -100000.0;
                    mbr_update(bench->meeting_buckets[slot].mbr, p1);
                    mbr_update(bench->meeting_buckets[slot].mbr, p2);

                    break;
                }
                slot = (slot + 1)%bench->config->num_meeting_buckets;
            }
        }
    }
}

__global__
void cuda_refinement_unroll(workbench *bench, uint offset){

	// the objects in which grid need be processed
	int loc = threadIdx.y;
	int pairid = blockIdx.x*blockDim.x+threadIdx.x;
	if(pairid>=bench->grid_check_counter){
		return;
	}

	uint gid = bench->grid_check[pairid].gid;

	uint size = min(bench->grid_counter[gid],bench->grid_capacity);
	if(loc+offset>=size){
		return;
	}
	uint pid = bench->grid_check[pairid].pid;
	uint target_pid = *(bench->grids+bench->grid_capacity*gid+offset+loc);
	if(!bench->grid_check[pairid].inside||pid<target_pid){
		double dist = distance(bench->points[pid].x, bench->points[pid].y, bench->points[target_pid].x, bench->points[target_pid].y);
		if(dist<=bench->config->reach_distance){
			uint pid1 = min(pid,target_pid);
			uint pid2 = max(target_pid,pid);
			size_t key = ((size_t)pid1+pid2)*(pid1+pid2+1)/2+pid2;
			size_t slot = key%bench->config->num_meeting_buckets;
			int ite = 0;
			while (ite++<5){
				unsigned long long prev = atomicCAS((unsigned long long *)&bench->meeting_buckets[slot].key, ULL_MAX, (unsigned long long)key);
				//printf("%ld\n",prev,ULL_MAX,bench->meeting_buckets[bench->current_bucket][slot].key);
				if(prev == key){
					bench->meeting_buckets[slot].end = bench->cur_time;
					break;
				}else if (prev == ULL_MAX){
					bench->meeting_buckets[slot].key = key;
					bench->meeting_buckets[slot].start = bench->cur_time;
					bench->meeting_buckets[slot].end = bench->cur_time;
					break;
				}
				slot = (slot + 1)%bench->config->num_meeting_buckets;
			}
		}
	}
}

/*
 * kernel function for identify completed meetings
 *
 * */

__global__
void cuda_profile_meetings(workbench *bench){

	size_t bid = blockIdx.x*blockDim.x+threadIdx.x;
	if(bid>=bench->config->num_meeting_buckets){
		return;
	}
	// empty
	if(bench->meeting_buckets[bid].key==ULL_MAX){
		return;
	}
	if(bench->config->profile){
		atomicAdd((unsigned long long *)&bench->num_taken_buckets, (unsigned long long)1);
	}
	// is still active
	if(bench->meeting_buckets[bid].end==bench->cur_time){
		if(bench->config->profile){
			atomicAdd((unsigned long long *)&bench->num_active_meetings, (unsigned long long)1);
		}
		return;
	}
}


__global__
void cuda_identify_meetings(workbench *bench) {
    size_t bid = blockIdx.x * blockDim.x + threadIdx.x;
    if (bid >= bench->config->num_meeting_buckets) {
        return;
    }
    // empty
    if (bench->meeting_buckets[bid].key == ULL_MAX) {
        return;
    }
    // is still active
    if (bench->meeting_buckets[bid].end == bench->cur_time) {
        if(bench->search_single) {
            if (bench->cur_time - bench->meeting_buckets[bid].start >= bench->config->min_meet_time + 1) {
                if (bench->search_single_pid == getpid1(bench->meeting_buckets[bid].key)) {
                    uint meeting_idx = atomicAdd(&bench->single_find_count, 1);
                    assert(bench->single_find_count < bench->config->search_single_capacity);
                    //bench->search_multi_list[meeting_idx].pid = bench->search_single_pid;
                    bench->search_single_list[meeting_idx].target = getpid2(bench->meeting_buckets[bid].key);
                    bench->search_single_list[meeting_idx].start = bench->meeting_buckets[bid].start;
                    bench->search_single_list[meeting_idx].end = bench->meeting_buckets[bid].end;                  //real end
                    bench->search_single_list[meeting_idx].low0 = bench->meeting_buckets[bid].mbr.low[0];
                    bench->search_single_list[meeting_idx].low1 = bench->meeting_buckets[bid].mbr.low[1];
                    bench->search_single_list[meeting_idx].high0 = bench->meeting_buckets[bid].mbr.high[0];
                    bench->search_single_list[meeting_idx].high1 = bench->meeting_buckets[bid].mbr.high[1];
                }
                else if (bench->search_single_pid == getpid2(bench->meeting_buckets[bid].key)) {
                    uint meeting_idx = atomicAdd(&bench->single_find_count, 1);
                    assert(bench->single_find_count < bench->config->search_single_capacity);
                    bench->search_single_list[meeting_idx].target = getpid1(bench->meeting_buckets[bid].key);
                    bench->search_single_list[meeting_idx].start = bench->meeting_buckets[bid].start;
                    bench->search_single_list[meeting_idx].end = bench->meeting_buckets[bid].end;                  //real end
                    bench->search_single_list[meeting_idx].low0 = bench->meeting_buckets[bid].mbr.low[0];
                    bench->search_single_list[meeting_idx].low1 = bench->meeting_buckets[bid].mbr.low[1];
                    bench->search_single_list[meeting_idx].high0 = bench->meeting_buckets[bid].mbr.high[0];
                    bench->search_single_list[meeting_idx].high1 = bench->meeting_buckets[bid].mbr.high[1];
                }
            }
        }
        if(bench->search_multi) {
            if (bench->cur_time - bench->meeting_buckets[bid].start >= bench->config->min_meet_time + 1) {
                for(int i = 0;i<bench->search_multi_length;i++){
                    if (bench->search_multi_pid[i] == getpid1(bench->meeting_buckets[bid].key)) {
                        uint meeting_idx = atomicAdd(&bench->multi_find_count, 1);
                        assert(bench->multi_find_count < bench->config->search_multi_capacity);
                        bench->search_multi_list[meeting_idx].pid = bench->search_multi_pid[i];
                        bench->search_multi_list[meeting_idx].target = getpid2(bench->meeting_buckets[bid].key);
                        bench->search_multi_list[meeting_idx].start = bench->meeting_buckets[bid].start;
                        bench->search_multi_list[meeting_idx].end = bench->meeting_buckets[bid].end;                  //real end
                        bench->search_multi_list[meeting_idx].low0 = bench->meeting_buckets[bid].mbr.low[0];
                        bench->search_multi_list[meeting_idx].low1 = bench->meeting_buckets[bid].mbr.low[1];
                        bench->search_multi_list[meeting_idx].high0 = bench->meeting_buckets[bid].mbr.high[0];
                        bench->search_multi_list[meeting_idx].high1 = bench->meeting_buckets[bid].mbr.high[1];
                    }
                    if (bench->search_multi_pid[i] == getpid2(bench->meeting_buckets[bid].key)) {
                        uint meeting_idx = atomicAdd(&bench->multi_find_count, 1);
                        assert(bench->multi_find_count < bench->config->search_multi_capacity);
                        bench->search_multi_list[meeting_idx].pid = bench->search_multi_pid[i];
                        bench->search_multi_list[meeting_idx].target = getpid1(bench->meeting_buckets[bid].key);
                        bench->search_multi_list[meeting_idx].start = bench->meeting_buckets[bid].start;
                        bench->search_multi_list[meeting_idx].end = bench->meeting_buckets[bid].end;                  //real end
                        bench->search_multi_list[meeting_idx].low0 = bench->meeting_buckets[bid].mbr.low[0];
                        bench->search_multi_list[meeting_idx].low1 = bench->meeting_buckets[bid].mbr.low[1];
                        bench->search_multi_list[meeting_idx].high0 = bench->meeting_buckets[bid].mbr.high[0];
                        bench->search_multi_list[meeting_idx].high1 = bench->meeting_buckets[bid].mbr.high[1];
                    }
                }
            }
        }
        return;
    }
    uint wid = 0;
    if (bench->cur_time - bench->meeting_buckets[bid].start >= bench->config->min_meet_time + 1) {
        if(bench->start_time_min > bench->meeting_buckets[bid].start){
            atomicMin(&bench->start_time_min,bench->meeting_buckets[bid].start);
        }
        if(bench->end_time_max < bench->meeting_buckets[bid].start){
            atomicMax(&bench->start_time_max,bench->meeting_buckets[bid].start);
        }

        uint pid, target;
        pid = getpid1(bench->meeting_buckets[bid].key);
        target = getpid2(bench->meeting_buckets[bid].key);

        uint low0 = (bench->meeting_buckets[bid].mbr.low[0] - bench->mbr.low[0])/(bench->mbr.high[0] - bench->mbr.low[0]) * 256;
        uint low1 = (bench->meeting_buckets[bid].mbr.low[1] - bench->mbr.low[1])/(bench->mbr.high[1] - bench->mbr.low[1]) * 256;
        uint high0 = (bench->meeting_buckets[bid].mbr.high[0] - bench->mbr.low[0])/(bench->mbr.high[0] - bench->mbr.low[0]) * 256;
        uint high1 = (bench->meeting_buckets[bid].mbr.high[1] - bench->mbr.low[1])/(bench->mbr.high[1] - bench->mbr.low[1]) * 256;

        uint s = (high0-low0+1)*(high1-low1+1);
        if(bench->kv_count<200){
            atomicAdd(&bench->s_of_all_mbr,s);
        }

        //printf("low0 %d,low1 %d,high0 %d,high1 %d",low0,low1,high0,high1);
        uint mid0 = (low0+high0)/2;
        uint mid1 = (low1+high1)/2;
        //printf("mid0 %d,mid1 %d",mid0,mid1);

        for (int k = 0; k < 2; k++) {
            uint meeting_idx = atomicAdd(&bench->kv_count, 1);
            assert(meeting_idx < bench->config->kv_capacity);
            if(k==1){
                uint swap = pid;
                pid = target;
                target = swap;
            }
            bench->d_keys[meeting_idx] = ((uint64_t)wid << 48) + ((uint64_t)pid << 23) + ((uint64_t)(bench->meeting_buckets[bid].end - bench->end_time_min) << 8) + ((uint64_t)bench->same_pid_count[pid]);          //64 = 16 + 25 + 15 + 8
            bench->d_values[meeting_idx] = ((__uint128_t)(bench->meeting_buckets[bid].end - bench->meeting_buckets[bid].start) << 113) + ((__uint128_t)target << 88) + box_to_128(&bench->meeting_buckets[bid].mbr);
            uint old_mid0 = 0;
            uint old_mid1 = 0;
            d2xy(bench->bitmap_edge_length,bench->d_wids[pid],old_mid0,old_mid1);
            uint new_mid0 = (old_mid0*bench->same_pid_count[pid] + mid0) / (bench->same_pid_count[pid] + 1);                //centroid
            uint new_mid1 = (old_mid1*bench->same_pid_count[pid] + mid1) / (bench->same_pid_count[pid] + 1);
            //printf("new_mid0 %d,new_mid1 %d",new_mid0,new_mid1);
            bench->d_wids[pid] = xy2d(bench->bitmap_edge_length,new_mid0, new_mid1);
            //printf("hilbert %d\n", bench->d_wids[pid]);

            bench->same_pid_count[pid]++;
        }
    }
    // reset the bucket
    bench->meeting_buckets[bid].key = ULL_MAX;
}

__global__
void cuda_search_single_kv(workbench *bench){
    uint kid = blockIdx.x*blockDim.x+threadIdx.x;
    if(kid>=bench->kv_count){
        return;
    }
    if((uint)(bench->d_keys[kid] >> 23 & ((1ULL << 25) - 1)) == bench->search_single_pid){              //all the same
        uint meeting_idx = atomicAdd(&bench->single_find_count, 1);
        assert(bench->single_find_count<bench->config->search_single_capacity);
        bench->search_single_list[meeting_idx].end = ((bench->d_keys[kid] >> 8) & ((1ULL << 15) - 1)) + bench->end_time_min;
        bench->search_single_list[meeting_idx].start = bench->search_single_list[meeting_idx].end - (bench->d_values[kid] >> 113);
        bench->search_single_list[meeting_idx].target = ((bench->d_values[kid] >> 88) & ((1ULL << 25) - 1));
        bench->search_single_list[meeting_idx].low0 = uint_to_float((uint)((bench->d_values[kid] >> 66) & ((1ULL << 22) - 1)));
        bench->search_single_list[meeting_idx].low1 = uint_to_float((uint)((bench->d_values[kid] >> 44) & ((1ULL << 22) - 1)));
        bench->search_single_list[meeting_idx].high0 = uint_to_float((uint)((bench->d_values[kid] >> 22) & ((1ULL << 22) - 1)));
        bench->search_single_list[meeting_idx].high1 = uint_to_float((uint)(bench->d_values[kid] & ((1ULL << 22) - 1)));
    }
}

__global__
void cuda_search_multi_kv(workbench *bench){
    uint kid = blockIdx.x*blockDim.x+threadIdx.x;
    if(kid>=bench->kv_count){
        return;
    }
    for(int i = 0;i<bench->search_multi_length;i++){
        if((uint)(bench->d_keys[kid] >> 23 & ((1ULL << 25) - 1)) == bench->search_multi_pid[i]){
            uint meeting_idx = atomicAdd(&bench->multi_find_count, 1);
            assert(bench->multi_find_count < bench->config->search_multi_capacity);
            bench->search_multi_list[meeting_idx].pid = bench->search_multi_pid[i];
            bench->search_multi_list[meeting_idx].end = ((bench->d_keys[kid] >> 8) & ((1ULL << 15) - 1)) + bench->end_time_min;
            bench->search_multi_list[meeting_idx].start = bench->search_multi_list[meeting_idx].end - (bench->d_values[kid] >> 113);
            bench->search_multi_list[meeting_idx].target = ((bench->d_values[kid] >> 88) & ((1ULL << 25) - 1));
            bench->search_multi_list[meeting_idx].low0 = uint_to_float((uint)((bench->d_values[kid] >> 66) & ((1ULL << 22) - 1)));
            bench->search_multi_list[meeting_idx].low1 = uint_to_float((uint)((bench->d_values[kid] >> 44) & ((1ULL << 22) - 1)));
            bench->search_multi_list[meeting_idx].high0 = uint_to_float((uint)((bench->d_values[kid] >> 22) & ((1ULL << 22) - 1)));
            bench->search_multi_list[meeting_idx].high1 = uint_to_float((uint)(bench->d_values[kid] & ((1ULL << 22) - 1)));
        }
    }
}

__global__
void write_wid(workbench *bench){
    uint kid = blockIdx.x*blockDim.x+threadIdx.x;
    if(kid>=bench->kv_count){
        return;
    }
    uint pid = (bench->d_keys[kid] >> 23) & ((1ULL << 25) - 1);
//    if(bench->d_wids[pid]>bench->bit_count){
//        printf("bench->d_wids[pid]%d bench->bit_count%d\n",bench->d_wids[pid],bench->bit_count);
//        uint x, y;
//        decodeZOrder(bench->d_wids[pid], x, y);
//        printf("x%d y%d\n",x,y);
//    }
    assert(bench->d_wids[pid]<=bench->bit_count);
    bench->d_keys[kid] += ((uint64_t)bench->d_wids[pid] << 48);
}

__global__
void BloomFilter_Add(workbench *bench){
    uint kid = blockIdx.x*blockDim.x+threadIdx.x;
    if(kid>=bench->config->kv_restriction){
        return;
    }

    uint pdwHashPos;
    uint64_t hash1, hash2;
    uint key = bench->d_keys[kid]/100000000 / 100000000 / 100000000;
    for(int i=0;i<bench->dwHashFuncs; i++){
        hash1 = d_MurmurHash2_x64((const void *)&key, sizeof(uint), bench->dwSeed);            // 双重散列封装，k个函数函数, 比如要20个
        hash2 = d_MurmurHash2_x64((const void *)&key, sizeof(uint), MIX_UINT64(hash1));
        pdwHashPos = (hash1 + i*hash2) % bench->dwFilterBits;
        bench->d_pstFilter[pdwHashPos/8] |= (1<<(pdwHashPos%8));
    }
}

__global__
void mbr_bitmap(workbench *bench){
    uint kid = blockIdx.x*blockDim.x+threadIdx.x;
    if(kid>=bench->config->kv_restriction){
        return;
    }
//    uint low0,low1,high0,high1;
//    float f_low0,f_low1,f_high0,f_high1;

    float f_low0 = uint_to_float((uint)((bench->d_values[kid] >> 66) & ((1ULL << 22) - 1)));
    float f_low1 = uint_to_float((uint)((bench->d_values[kid] >> 44) & ((1ULL << 22) - 1)));
    float f_high0 = uint_to_float((uint)((bench->d_values[kid] >> 22) & ((1ULL << 22) - 1)));
    float f_high1 = uint_to_float((uint)(bench->d_values[kid] & ((1ULL << 22) - 1)));

    uint low0 = (f_low0 - bench->mbr.low[0])/(bench->mbr.high[0] - bench->mbr.low[0]) * 256;
    uint low1 = (f_low1 - bench->mbr.low[1])/(bench->mbr.high[1] - bench->mbr.low[1]) * 256;
    uint high0 = (f_high0 - bench->mbr.low[0])/(bench->mbr.high[0] - bench->mbr.low[0]) * 256;
    uint high1 = (f_high1 - bench->mbr.low[1])/(bench->mbr.high[1] - bench->mbr.low[1]) * 256;
    //4
    //uint bitmap_id = kid/(bench->bit_count/8);          //256*256/8 = 8192B 1B=1char    ???????
    uint bitmap_id = kid/(bench->config->kv_restriction / bench->config->SSTable_count);           //kid/65536
    uint bit_pos = 0;
    for(uint i=low0;i<=high0;i++){
        for(uint j=low1;j<=high1;j++){
            bit_pos = xy2d(bench->bitmap_edge_length,i,j);
            bench->d_bitmaps[bitmap_id*(bench->bit_count/8)+bit_pos/8] |= (1<<(bit_pos%8));
        }
    }
}

__global__
void wid_bitmap(workbench *bench){
    uint kid = blockIdx.x*blockDim.x+threadIdx.x;
    if(kid>=bench->config->kv_restriction){
        return;
    }
    uint wid = bench->d_keys[kid] >> 48;
    uint bitmap_id = kid/(bench->config->kv_restriction / bench->config->SSTable_count);           //kid/65536
    bench->d_bitmaps[bitmap_id*(bench->bit_count/8)+wid/8] |= (1<<(wid%8));
}

/*
 * kernel functions for index update
 *
 * */
__global__
void cuda_update_schema_split(workbench *bench, uint size){
	uint sidx = blockIdx.x*blockDim.x+threadIdx.x;
	if(sidx>=size){
		return;
	}
	uint curnode = bench->split_list[sidx];
	//printf("split: %d\n",curnode);
	//schema[curnode].mbr.print();
	bench->schema[curnode].type = BRANCH;
	// reuse by one of its child
	uint gid = bench->schema[curnode].grid_id;

	double xhalf = bench->schema[curnode].mid_x-bench->schema[curnode].mbr.low[0];
	double yhalf = bench->schema[curnode].mid_y-bench->schema[curnode].mbr.low[1];

	for(int i=0;i<4;i++){
		// pop space for schema and grid
		uint idx = atomicAdd(&bench->schema_stack_index, 1);
		assert(idx<bench->schema_stack_capacity);
		uint child = bench->schema_stack[idx];
		//printf("sidx: %d %d\n",idx,child);
		bench->schema[curnode].children[i] = child;

		if(i>0){
			idx = atomicAdd(&bench->grids_stack_index,1);
			assert(idx<bench->grids_stack_capacity);
			gid = bench->grids_stack[idx];
			//printf("gidx: %d %d\n",idx,gid);
		}
		bench->schema[child].grid_id = gid;
		bench->grid_counter[gid] = 0;
		bench->schema[child].level = bench->schema[curnode].level+1;
		bench->schema[child].type = LEAF;
		bench->schema[child].overflow_count = 0;
		bench->schema[child].underflow_count = 0;

		bench->schema[child].mbr.low[0] = bench->schema[curnode].mbr.low[0]+(i%2==1)*xhalf;
		bench->schema[child].mbr.low[1] = bench->schema[curnode].mbr.low[1]+(i/2==1)*yhalf;
		bench->schema[child].mbr.high[0] = bench->schema[curnode].mid_x+(i%2==1)*xhalf;
		bench->schema[child].mbr.high[1] = bench->schema[curnode].mid_y+(i/2==1)*yhalf;
		bench->schema[child].mid_x = (bench->schema[child].mbr.low[0]+bench->schema[child].mbr.high[0])/2;
		bench->schema[child].mid_y = (bench->schema[child].mbr.low[1]+bench->schema[child].mbr.high[1])/2;
	}
}
__global__
void cuda_update_schema_merge(workbench *bench, uint size){
	uint sidx = blockIdx.x*blockDim.x+threadIdx.x;
	if(sidx>=size){
		return;
	}
	uint curnode = bench->merge_list[sidx];
	//reclaim the children
	uint gid = 0;
	for(int i=0;i<4;i++){
		uint child_offset = bench->schema[curnode].children[i];
		assert(bench->schema[child_offset].type==LEAF);
		//bench->schema[child_offset].mbr.print();
		// push the bench->schema and grid spaces to stack for reuse

		bench->grid_counter[bench->schema[child_offset].grid_id] = 0;
		if(i<3){
			// push to stack
			uint idx = atomicSub(&bench->grids_stack_index,1)-1;
			bench->grids_stack[idx] = bench->schema[child_offset].grid_id;
		}else{
			// reused by curnode
			gid = bench->schema[child_offset].grid_id;
		}
		bench->schema[child_offset].type = INVALID;
		uint idx = atomicSub(&bench->schema_stack_index,1)-1;
		bench->schema_stack[idx] = child_offset;
	}
	bench->schema[curnode].type = LEAF;
	// reuse the grid of one of its child
	bench->schema[curnode].grid_id = gid;
}

__global__
void cuda_update_schema_collect(workbench *bench){
	uint curnode = blockIdx.x*blockDim.x+threadIdx.x;
	if(curnode>=bench->schema_stack_capacity){
		return;
	}
	if(bench->schema[curnode].type==LEAF){
		if(height(&bench->schema[curnode].mbr)>2*bench->config->reach_distance&&
				bench->grid_counter[bench->schema[curnode].grid_id]>bench->config->grid_capacity){
			// this node is overflowed a continuous number of times, split it
			if(++bench->schema[curnode].overflow_count>=bench->config->schema_update_delay){
				uint sidx = atomicAdd(&bench->split_list_index,1);
				bench->split_list[sidx] = curnode;
				bench->schema[curnode].overflow_count = 0;
			}
		}else{
			bench->schema[curnode].overflow_count = 0;
		}
	}else if(bench->schema[curnode].type==BRANCH){
		int leafchild = 0;
		int ncounter = 0;
		for(int i=0;i<4;i++){
			uint child_node = bench->schema[curnode].children[i];
			if(bench->schema[child_node].type==LEAF){
				leafchild++;
				ncounter += bench->grid_counter[bench->schema[child_node].grid_id];
			}
		}
		// this one need update
		if(leafchild==4&&ncounter<bench->config->grid_capacity){
			// this node need be merged
			if(++bench->schema[curnode].underflow_count>=bench->config->schema_update_delay){
				//printf("%d\n",curnode);
				uint sidx = atomicAdd(&bench->merge_list_index,1);
				bench->merge_list[sidx] = curnode;
				bench->schema[curnode].underflow_count = 0;
			}
		}else{
			bench->schema[curnode].underflow_count = 0;
		}
	}
}


__global__
void cuda_init_schema_stack(workbench *bench){
	uint curnode = blockIdx.x*blockDim.x+threadIdx.x;
	if(curnode>=bench->schema_stack_capacity){
		return;
	}
	bench->schema_stack[curnode] = curnode;
}
__global__
void cuda_init_grids_stack(workbench *bench){
	uint curnode = blockIdx.x*blockDim.x+threadIdx.x;
	if(curnode>=bench->grids_stack_capacity){
		return;
	}
	bench->grids_stack[curnode] = curnode;
}

#define one_dim 16384
//#define one_dim 8

__global__
void cuda_build_qtree(workbench *bench){
	uint pid = blockIdx.x*blockDim.x+threadIdx.x;
	if(pid>=bench->config->num_objects){
		return;
	}
	uint x = (bench->points[pid].x-bench->mbr.low[0])/(bench->mbr.high[0]-bench->mbr.low[0])*one_dim;
	uint y = (bench->points[pid].y-bench->mbr.low[1])/(bench->mbr.high[1]-bench->mbr.low[1])*one_dim;
	atomicAdd(&bench->part_counter[x+y*one_dim],1);
}

__global__
void cuda_clean_cells(workbench *bench){
	uint pid = blockIdx.x*blockDim.x+threadIdx.x;
	if(pid>=one_dim*one_dim){
		return;
	}
	bench->schema_assigned[pid] = 0;
	bench->part_counter[pid] = 0;
	if(pid==0){
		bench->grids_stack_index = 0;
		bench->schema_stack_index = 1;
	}
}

__global__
void cuda_merge_qtree(workbench *bench, uint gap){
	uint pid = blockIdx.x*blockDim.x+threadIdx.x;
	uint xdim = one_dim/gap;
	if(pid>=(xdim*xdim)){
		return;
	}

	uint x = pid%xdim;
	uint y = pid/xdim;
	if(gap==1){
		if(bench->part_counter[pid]>bench->config->grid_capacity){
			uint node = atomicAdd(&bench->schema_stack_index,1);
			bench->schema[node].type = BRANCH;
			bench->schema[node].mbr.low[0] = bench->mbr.low[0]+x*(bench->mbr.high[0]-bench->mbr.low[0])/xdim;
			bench->schema[node].mbr.low[1] = bench->mbr.low[1]+y*(bench->mbr.high[1]-bench->mbr.low[1])/xdim;
			bench->schema[node].mbr.high[0] = bench->mbr.low[0]+(x+1)*(bench->mbr.high[0]-bench->mbr.low[0])/xdim;
			bench->schema[node].mbr.high[1] = bench->mbr.low[1]+(y+1)*(bench->mbr.high[1]-bench->mbr.low[1])/xdim;
			bench->schema[node].mid_x = (bench->schema[node].mbr.low[0]+bench->schema[node].mbr.high[0])/2;
			bench->schema[node].mid_y = (bench->schema[node].mbr.low[1]+bench->schema[node].mbr.high[1])/2;
			double xhalf = bench->schema[node].mid_x-bench->schema[node].mbr.low[0];
			double yhalf = bench->schema[node].mid_y-bench->schema[node].mbr.low[1];

			for(uint i=0;i<4;i++){
				uint cnode = atomicAdd(&bench->schema_stack_index,1);
				bench->schema[cnode].grid_id = atomicAdd(&bench->grids_stack_index,1);
				bench->schema[cnode].type = LEAF;
				bench->schema[node].children[i] = cnode;
				bench->grid_counter[bench->schema[cnode].grid_id] = 0;
				bench->schema[cnode].type = LEAF;
				bench->schema[cnode].overflow_count = 0;
				bench->schema[cnode].underflow_count = 0;
				bench->schema[cnode].mbr.low[0] = bench->schema[node].mbr.low[0]+(i%2==1)*xhalf;
				bench->schema[cnode].mbr.low[1] = bench->schema[node].mbr.low[1]+(i/2==1)*yhalf;
				bench->schema[cnode].mbr.high[0] = bench->schema[node].mid_x+(i%2==1)*xhalf;
				bench->schema[cnode].mbr.high[1] = bench->schema[node].mid_y+(i/2==1)*yhalf;
				bench->schema[cnode].mid_x = (bench->schema[cnode].mbr.low[0]+bench->schema[cnode].mbr.high[0])/2;
				bench->schema[cnode].mid_y = (bench->schema[cnode].mbr.low[1]+bench->schema[cnode].mbr.high[1])/2;
				//print_box(&bench->schema[cnode].mbr);
			}
			bench->schema_assigned[pid] = node;
		}
	}else{

		uint step = gap/2;
		uint p[4];
		p[0] = y*gap*one_dim+x*gap;
		p[1] = y*gap*one_dim+x*gap+step;
		p[2] = y*gap*one_dim+step*one_dim+x*gap;
		p[3] = y*gap*one_dim+step*one_dim+x*gap+step;
		uint size = 0;
		for(uint i=0;i<4;i++){
			size += bench->part_counter[p[i]];
		}
		// parent node
		if(size>bench->config->grid_capacity){
			uint node = 0;
			// node 0 is for the root only
			if(xdim!=1){
				node = atomicAdd(&bench->schema_stack_index,1);
			}
			bench->schema[node].type = BRANCH;
			bench->schema[node].mbr.low[0] = bench->mbr.low[0]+x*(bench->mbr.high[0]-bench->mbr.low[0])/xdim;
			bench->schema[node].mbr.low[1] = bench->mbr.low[1]+y*(bench->mbr.high[1]-bench->mbr.low[1])/xdim;
			bench->schema[node].mbr.high[0] = bench->mbr.low[0]+(x+1)*(bench->mbr.high[0]-bench->mbr.low[0])/xdim;
			bench->schema[node].mbr.high[1] = bench->mbr.low[1]+(y+1)*(bench->mbr.high[1]-bench->mbr.low[1])/xdim;
			bench->schema[node].mid_x = (bench->schema[node].mbr.low[0]+bench->schema[node].mbr.high[0])/2;
			bench->schema[node].mid_y = (bench->schema[node].mbr.low[1]+bench->schema[node].mbr.high[1])/2;

			double xhalf = bench->schema[node].mid_x-bench->schema[node].mbr.low[0];
			double yhalf = bench->schema[node].mid_y-bench->schema[node].mbr.low[1];
			for(uint i=0;i<4;i++){
				uint cnode = 0;
				if(bench->schema_assigned[p[i]]!=0){
					cnode = bench->schema_assigned[p[i]];
				}else{
					cnode = atomicAdd(&bench->schema_stack_index,1);
					bench->schema[cnode].grid_id = atomicAdd(&bench->grids_stack_index,1);
					bench->schema[cnode].type = LEAF;
					bench->grid_counter[bench->schema[cnode].grid_id] = 0;
					bench->schema[cnode].type = LEAF;
					bench->schema[cnode].overflow_count = 0;
					bench->schema[cnode].underflow_count = 0;
					bench->schema[cnode].mbr.low[0] = bench->schema[node].mbr.low[0]+(i%2==1)*xhalf;
					bench->schema[cnode].mbr.low[1] = bench->schema[node].mbr.low[1]+(i/2==1)*yhalf;
					bench->schema[cnode].mbr.high[0] = bench->schema[node].mid_x+(i%2==1)*xhalf;
					bench->schema[cnode].mbr.high[1] = bench->schema[node].mid_y+(i/2==1)*yhalf;
					bench->schema[cnode].mid_x = (bench->schema[cnode].mbr.low[0]+bench->schema[cnode].mbr.high[0])/2;
					bench->schema[cnode].mid_y = (bench->schema[cnode].mbr.low[1]+bench->schema[cnode].mbr.high[1])/2;
					//print_box(&bench->schema[cnode].mbr);
				}
				bench->schema[node].children[i] = cnode;
			}
			bench->schema_assigned[p[0]] = node;
		}
		// for next upper level
		bench->part_counter[p[0]] = size;
	}
}

workbench *cuda_create_device_bench(workbench *bench, gpu_info *gpu){
	log("GPU memory:");
	struct timeval start = get_cur_time();
	gpu->clear();
	// use h_bench as a container to copy in and out GPU
	workbench h_bench(bench);
	// space for the raw points data
	h_bench.points = (Point *)gpu->allocate(bench->config->num_objects*sizeof(Point));
	size_t size = bench->config->num_objects*sizeof(Point);
	log("\t%.2f MB\tpoints",1.0*size/1024/1024);

	// space for the pids of all the grids
	h_bench.grids = (uint *)gpu->allocate(bench->grids_stack_capacity*bench->grid_capacity*sizeof(uint));
	h_bench.grid_counter = (uint *)gpu->allocate(bench->grids_stack_capacity*sizeof(uint));
	h_bench.grids_stack = (uint *)gpu->allocate(bench->grids_stack_capacity*sizeof(uint));
	size = bench->grids_stack_capacity*bench->grid_capacity*sizeof(uint)+bench->grids_stack_capacity*sizeof(uint)+bench->grids_stack_capacity*sizeof(uint);
	log("\t%.2f MB\tgrids",1.0*size/1024/1024);

	// space for the QTtree schema
	h_bench.schema = (QTSchema *)gpu->allocate(bench->schema_stack_capacity*sizeof(QTSchema));
	h_bench.schema_stack = (uint *)gpu->allocate(bench->schema_stack_capacity*sizeof(uint));
	size = bench->schema_stack_capacity*sizeof(QTSchema)+bench->schema_stack_capacity*sizeof(uint);
	log("\t%.2f MB\tschema",1.0*size/1024/1024);

	// space for the pid-zid pairs
	h_bench.grid_check = (checking_unit *)gpu->allocate(bench->grid_check_capacity*sizeof(checking_unit));
	size = bench->grid_check_capacity*sizeof(checking_unit);
	log("\t%.2f MB\trefine list",1.0*size/1024/1024);


	size = 2*bench->filter_list_capacity*sizeof(uint);
	h_bench.filter_list = (uint *)gpu->allocate(size);
	log("\t%.2f MB\tfiltering list",1.0*size/1024/1024);


	// space for processing stack
	h_bench.tmp_space = (uint *)gpu->allocate(bench->tmp_space_capacity*sizeof(uint));
	size = bench->tmp_space_capacity*sizeof(uint);
	h_bench.merge_list = h_bench.tmp_space;
	h_bench.split_list = h_bench.tmp_space+bench->tmp_space_capacity/2;
	log("\t%.2f MB\ttemporary space",1.0*size/1024/1024);

	h_bench.meeting_buckets = (meeting_unit *)gpu->allocate(bench->config->num_meeting_buckets*sizeof(meeting_unit));
	size = bench->config->num_meeting_buckets*sizeof(meeting_unit);
	log("\t%.2f MB\thash table",1.0*size/1024/1024);

//	h_bench.meetings = (meeting_unit *)gpu->allocate(bench->meeting_capacity*sizeof(meeting_unit));
//	size = bench->meeting_capacity*sizeof(meeting_unit);
//	log("\t%.2f MB\tmeetings",1.0*size/1024/1024);

    //cuda sort
    h_bench.d_keys = (uint64_t *)gpu->allocate(bench->config->kv_capacity*sizeof(uint64_t));
    size = bench->config->kv_capacity*sizeof(uint64_t);
    log("\t%.2f MB\td_keys",1.0*size/1024/1024);
    h_bench.d_values = (__uint128_t *)gpu->allocate(bench->config->kv_capacity*sizeof(__uint128_t));
    size = bench->config->kv_capacity*sizeof(__uint128_t);
    log("\t%.2f MB\td_values",1.0*size/1024/1024);

    //cuda search
    h_bench.search_single_list = (search_info_unit *)gpu->allocate(bench->config->search_single_capacity*sizeof(search_info_unit));
    size = bench->config->search_single_capacity*sizeof(search_info_unit);
    log("\t%.2f MB\tsearch_single_list",1.0*size/1024/1024);
    h_bench.search_multi_pid = (uint *)gpu->allocate(bench->config->search_single_capacity*sizeof(uint));
    size = bench->config->search_single_capacity*sizeof(uint);
    log("\t%.2f MB\tsearch_single_list",1.0*size/1024/1024);
    h_bench.search_multi_list = (search_info_unit *)gpu->allocate(bench->config->search_multi_capacity*sizeof(search_info_unit));
    size = bench->config->search_single_capacity*sizeof(search_info_unit);
    log("\t%.2f MB\tsearch_single_list",1.0*size/1024/1024);

    if(bench->config->bloom_filter) {
        //bloom filter
        h_bench.d_pstFilter = (unsigned char *) gpu->allocate(bench->dwFilterSize);
        size = bench->dwFilterSize;
        log("\t%.2f MB\td_pstFilter", 1.0 * size / 1024 / 1024);
        hipMemset(h_bench.d_pstFilter, 0, bench->dwFilterSize);
    }

    //bitmap
    if(true) {
        //bloom filter
        h_bench.d_bitmaps = (unsigned char *) gpu->allocate(bench->bitmaps_size);
        size = bench->bitmaps_size;
        log("\t%.2f MB\td_bitmaps", 1.0 * size / 1024 / 1024);
        hipMemset(h_bench.d_bitmaps, 0, size);

        h_bench.d_wids = (unsigned short*)gpu->allocate(bench->config->num_objects*sizeof(unsigned short));
        size = bench->config->num_objects*sizeof(unsigned short);
        log("\t%.2f MB\td_wids", 1.0 * size / 1024 / 1024);

        h_bench.same_pid_count = (unsigned short *)gpu->allocate(bench->config->num_objects * sizeof(unsigned short));
        size = bench->config->num_objects*sizeof(unsigned short);
        log("\t%.2f MB\tsame_name_count", 1.0 * size / 1024 / 1024);
    }

	h_bench.part_counter = (uint *)gpu->allocate(one_dim*one_dim*sizeof(uint));
	h_bench.schema_assigned = (uint *)gpu->allocate(one_dim*one_dim*sizeof(uint));

	// space for the configuration
	h_bench.config = (configuration *)gpu->allocate(sizeof(configuration));
	// space for the mapping of bench in GPU
	workbench *d_bench = (workbench *)gpu->allocate(sizeof(workbench));

	// the configuration and schema are fixed
	CUDA_SAFE_CALL(hipMemcpy(h_bench.schema, bench->schema, bench->schema_stack_capacity*sizeof(QTSchema), hipMemcpyHostToDevice));
	CUDA_SAFE_CALL(hipMemcpy(h_bench.config, bench->config, sizeof(configuration), hipMemcpyHostToDevice));
	CUDA_SAFE_CALL(hipMemcpy(d_bench, &h_bench, sizeof(workbench), hipMemcpyHostToDevice));

	cuda_init_grids_stack<<<bench->grids_stack_capacity/1024, 1024>>>(d_bench);
	cuda_init_schema_stack<<<bench->schema_stack_capacity/1024, 1024>>>(d_bench);
	cuda_clean_buckets<<<bench->config->num_meeting_buckets/1024+1,1024>>>(d_bench);

	logt("GPU allocating space %ld MB", start,gpu->size_allocated()/1024/1024);

	return d_bench;
}

/*
 *
 * check the reachability of objects in a list of partitions
 * ctx.data contains the list of
 *
 * */
void process_with_gpu(workbench *bench, workbench* d_bench, gpu_info *gpu){
	struct timeval start = get_cur_time();
	//gpu->print();
	assert(bench);
	assert(d_bench);
	assert(gpu);
	hipSetDevice(gpu->device_id);

	/* 1. copy data */
	// setup the current time and points for this round
	workbench h_bench(bench);
	CUDA_SAFE_CALL(hipMemcpy(&h_bench, d_bench, sizeof(workbench), hipMemcpyDeviceToHost));
    if(bench->search_single) {
        h_bench.search_single = true;
        h_bench.single_find_count = 0;
    }
    else {
        h_bench.search_single = false;
    }
    if(bench->search_multi) {
        h_bench.search_multi = true;
        h_bench.multi_find_count = 0;
        h_bench.search_multi_length = bench->search_multi_length;
        CUDA_SAFE_CALL(hipMemcpy(h_bench.search_multi_pid, bench->search_multi_pid, bench->search_multi_length * sizeof(search_info_unit), hipMemcpyHostToDevice));
    }
    else {
        h_bench.search_multi = false;
    }
	h_bench.cur_time = bench->cur_time;
    h_bench.end_time_min = bench->end_time_min;
    h_bench.s_of_all_mbr = 0;
	CUDA_SAFE_CALL(hipMemcpy(d_bench, &h_bench, sizeof(workbench), hipMemcpyHostToDevice));
	CUDA_SAFE_CALL(hipMemcpy(h_bench.points, bench->points, bench->config->num_objects*sizeof(Point), hipMemcpyHostToDevice));
	bench->pro.copy_time += get_time_elapsed(start,false);
	logt("copy in data", start);

	if(!bench->config->dynamic_schema){
		struct timeval newstart = get_cur_time();
		cuda_clean_cells<<<one_dim*one_dim/1024+1,1024>>>(d_bench);
		cuda_build_qtree<<<bench->config->num_objects/1024+1,1024>>>(d_bench);
//		check_execution();
//		hipDeviceSynchronize();
//		logt("build qtree", newstart);

		for(uint i=1;i<=one_dim;i*=2){
			uint num = one_dim*one_dim/(i*i);
			cuda_merge_qtree<<<num/1024+1,1024>>>(d_bench,i);
//			check_execution();
//			hipDeviceSynchronize();
//			CUDA_SAFE_CALL(hipMemcpy(&h_bench, d_bench, sizeof(workbench), hipMemcpyDeviceToHost));
//			logt("merge qtree %d %d %d %d", newstart,i, h_bench.schema_stack_index, h_bench.grids_stack_index, h_bench.grid_check_counter);
		}
		CUDA_SAFE_CALL(hipMemcpy(&h_bench, d_bench, sizeof(workbench), hipMemcpyDeviceToHost));
		hipDeviceSynchronize();
		bench->pro.index_update_time += get_time_elapsed(start,false);
		logt("build qtree %d nodes %d partitions", start, h_bench.schema_stack_index, h_bench.grids_stack_index);
		//exit(0);
	}

	/* 2. filtering */
	if(bench->config->phased_lookup){
		// do the partition
		cuda_partition<<<bench->config->num_objects/1024+1,1024>>>(d_bench);

		CUDA_SAFE_CALL(hipMemcpy(&h_bench, d_bench, sizeof(workbench), hipMemcpyDeviceToHost));
		bench->pro.partition_time += get_time_elapsed(start,false);
		logt("partition data %d still need lookup", start,h_bench.filter_list_index);
		bench->filter_list_index = h_bench.filter_list_index;
	}else{
		cuda_pack_lookup<<<bench->config->num_objects/1024+1,1024>>>(d_bench);
		check_execution();
		hipDeviceSynchronize();
		CUDA_SAFE_CALL(hipMemcpy(&h_bench, d_bench, sizeof(workbench), hipMemcpyDeviceToHost));
	}

	uint batch_size = bench->tmp_space_capacity/(PER_STACK_SIZE*2+1);
	for(int i=0;i<h_bench.filter_list_index;i+=batch_size){
		int bs = min(batch_size,h_bench.filter_list_index-i);
		cuda_filtering<<<bs/1024+1,1024>>>(d_bench, i, bs, !bench->config->phased_lookup);
		check_execution();
		hipDeviceSynchronize();
	}
	CUDA_SAFE_CALL(hipMemcpy(&h_bench, d_bench, sizeof(workbench), hipMemcpyDeviceToHost));
	bench->pro.filter_time += get_time_elapsed(start,false);
	logt("filtering with %d checkings", start,h_bench.grid_check_counter);

	/* 3. refinement step */
	if(false){
		for(uint offset=0;offset<bench->grid_capacity;offset+=bench->config->zone_capacity){
			struct timeval ss = get_cur_time();
			bench->grid_check_counter = h_bench.grid_check_counter;
			uint thread_y = bench->config->zone_capacity;
			uint thread_x = 1024/thread_y;
			dim3 block(thread_x, thread_y);
			cuda_refinement_unroll<<<h_bench.grid_check_counter/thread_x+1,block>>>(d_bench,offset);
			check_execution();
			hipDeviceSynchronize();
			logt("process %d",ss,offset);
		}
		CUDA_SAFE_CALL(hipMemcpy(&h_bench, d_bench, sizeof(workbench), hipMemcpyDeviceToHost));
		bench->pro.refine_time += get_time_elapsed(start,false);
		logt("refinement step", start);
	}else{
		if(bench->config->unroll){
			cuda_unroll<<<h_bench.grid_check_counter/1024+1,1024>>>(d_bench,h_bench.grid_check_counter);
			check_execution();
			hipDeviceSynchronize();
			CUDA_SAFE_CALL(hipMemcpy(&h_bench, d_bench, sizeof(workbench), hipMemcpyDeviceToHost));
			bench->pro.refine_time += get_time_elapsed(start,false);
			logt("%d pid-grid-offset tuples need be checked", start,h_bench.grid_check_counter);
		}

		bench->grid_check_counter = h_bench.grid_check_counter;
		uint thread_y = bench->config->unroll?bench->config->zone_capacity:bench->grid_capacity;
		uint thread_x = 1024/thread_y;
		dim3 block(thread_x, thread_y);
		cuda_refinement<<<h_bench.grid_check_counter/thread_x+1,block>>>(d_bench);
		check_execution();
		hipDeviceSynchronize();
		CUDA_SAFE_CALL(hipMemcpy(&h_bench, d_bench, sizeof(workbench), hipMemcpyDeviceToHost));
		bench->pro.refine_time += get_time_elapsed(start,false);
		logt("refinement step", start);
	}

	/* 4. identify the completed meetings */
	if(bench->config->profile){
		cuda_profile_meetings<<<bench->config->num_meeting_buckets/1024+1,1024>>>(d_bench);
		check_execution();
		hipDeviceSynchronize();
		logt("profile meetings",start);
	}
    int before_kv = h_bench.kv_count;
	cuda_identify_meetings<<<bench->config->num_meeting_buckets/1024+1,1024>>>(d_bench);
	check_execution();
	hipDeviceSynchronize();
	CUDA_SAFE_CALL(hipMemcpy(&h_bench, d_bench, sizeof(workbench), hipMemcpyDeviceToHost));
    bench->pro.meeting_identify_time += get_time_elapsed(start,false);
    int kv_increase = h_bench.kv_count-before_kv;
    printf("second %d finished meetings : %d\n",bench->cur_time , kv_increase);
    //cout<<"ave_s_mbr"<<(float)h_bench.s_of_all_mbr/100<<endl;
    //h_bench.s_of_all_mbr = 0;
    logt("meeting identify: %d meetings", start, kv_increase);
	bench->num_active_meetings = h_bench.num_active_meetings;
	bench->num_taken_buckets = h_bench.num_taken_buckets;
    //bench->kv_count = h_bench.kv_count;
	//logt("meeting identify: %d taken %d active %d new meetings found", start, h_bench.num_taken_buckets, h_bench.num_active_meetings, h_bench.meeting_counter);

    //4.5 cuda sort
    if(h_bench.kv_count>bench->config->kv_restriction){
        bench->start_time_min = h_bench.start_time_min;
        bench->start_time_max = h_bench.start_time_max;
        h_bench.start_time_min = (1ULL<<32) -1;
        h_bench.start_time_max = 0;
        uint offset = 0;
        if(bench->big_sorted_run_count%2==1){
            offset = bench->config->MemTable_capacity/2;
        }
        if(true){
            write_wid<<<h_bench.kv_count / 1024 + 1,1024>>>(d_bench);
            check_execution();
            hipDeviceSynchronize();
            CUDA_SAFE_CALL(hipMemcpy(&h_bench, d_bench, sizeof(workbench), hipMemcpyDeviceToHost));
            //logt("bloom filter ", start);
            CUDA_SAFE_CALL(hipMemcpy(bench->h_wids[offset+bench->MemTable_count], h_bench.d_wids, bench->config->num_objects*sizeof(unsigned short), hipMemcpyDeviceToHost));

//            hipMemset(h_bench.d_wids, 0, bench->config->num_objects*sizeof(unsigned short));
//            hipMemset(h_bench.same_pid_count, 0, bench->config->num_objects * sizeof(unsigned short));
        }

        // wrap raw pointer with a device_ptr
        thrust::device_ptr<uint64_t> d_vector_keys = thrust::device_pointer_cast(h_bench.d_keys);
        thrust::device_ptr<__uint128_t> d_vector_values = thrust::device_pointer_cast(h_bench.d_values);
        bench->pro.cuda_sort_time += get_time_elapsed(start,false);
        logt("pointer_cast: ",start);
        // use device_ptr in Thrust algorithms
        thrust::sort_by_key(d_vector_keys, d_vector_keys + bench->config->kv_restriction, d_vector_values);
        check_execution();
        hipDeviceSynchronize();
        CUDA_SAFE_CALL(hipMemcpy(&h_bench, d_bench, sizeof(workbench), hipMemcpyDeviceToHost));
        bench->pro.cuda_sort_time += get_time_elapsed(start,false);
        logt("cuda_sort_time: ",start);
        CUDA_SAFE_CALL(hipMemcpy(bench->h_keys[offset+bench->MemTable_count], h_bench.d_keys, bench->config->kv_restriction * sizeof(uint64_t), hipMemcpyDeviceToHost));
        CUDA_SAFE_CALL(hipMemcpy(bench->h_values[offset+bench->MemTable_count], h_bench.d_values, bench->config->kv_restriction * sizeof(__uint128_t), hipMemcpyDeviceToHost));
        bench->pro.cuda_sort_time += get_time_elapsed(start,false);
        logt("hipMemcpy kv",start);

        //thrust::copy(d_vector_keys, d_vector_keys+bench->kv_count, bench->h_keys[bench->MemTable_count]);     //wrong
        //::copy(d_vector_values, d_vector_values+bench->kv_count, bench->h_values[bench->MemTable_count]);

        //bloom filter //bloom filter is useless, so code is not updated.

        print_128(bench->h_keys[offset+bench->MemTable_count][10]);
        printf("\n");
        print_128(bench->h_values[offset+bench->MemTable_count][10]);
        printf("\n");
        printf("hipMemcpy kv right\n");
        cout<<"bench->end_time_min:"<<bench->end_time_min<<endl;
        cout<<"wid:"<<(uint)(bench->h_keys[offset+bench->MemTable_count][10] >> 48)<<endl;
        cout<<"pid:"<<(uint)((bench->h_keys[offset+bench->MemTable_count][10] >> 23) & ((1ULL << 25) - 1))<<endl;
        cout<<"end:"<<(uint)((bench->h_keys[offset+bench->MemTable_count][10] >> 8) & ((1ULL << 15) - 1)) + bench->end_time_min<<endl;
        cout<<"count:"<<(uint)(bench->h_keys[offset+bench->MemTable_count][10] & ((1ULL << 8) - 1))<<endl;
        printf("\n");
        cout<<"duration:"<<(uint)(bench->h_values[offset+bench->MemTable_count][10] >> 113)<<endl;
        cout<<"target:"<<(uint)((bench->h_values[offset+bench->MemTable_count][10] >> 88) & ((1ULL << 25) - 1))<<endl;
        box temp_box(bench->h_values[offset+bench->MemTable_count][10]);
        temp_box.print();

        if(true){           //mbr bit map
            cout<<"h_bench.bit_count:"<<h_bench.bit_count<<endl;
            cout<<"h_bench.bitmaps_size:"<<h_bench.bitmaps_size<<endl;
            mbr_bitmap<<<bench->config->kv_restriction / 1024 + 1,1024>>>(d_bench);
            check_execution();
            hipDeviceSynchronize();
            CUDA_SAFE_CALL(hipMemcpy(&h_bench, d_bench, sizeof(workbench), hipMemcpyDeviceToHost));
            //logt("bloom filter ", start);
            CUDA_SAFE_CALL(hipMemcpy(bench->h_bitmaps[offset+bench->MemTable_count], h_bench.d_bitmaps, bench->bitmaps_size, hipMemcpyDeviceToHost));
            hipMemset(h_bench.d_bitmaps, 0, bench->bitmaps_size);
        }

//        if(true){           //wid bit map
//            wid_bitmap<<<bench->config->kv_restriction / 1024 + 1,1024>>>(d_bench);
//            check_execution();
//            hipDeviceSynchronize();
//            CUDA_SAFE_CALL(hipMemcpy(&h_bench, d_bench, sizeof(workbench), hipMemcpyDeviceToHost));
//            //logt("bloom filter ", start);
//            CUDA_SAFE_CALL(hipMemcpy(bench->h_bitmaps[offset+bench->MemTable_count], h_bench.d_bitmaps, bench->bitmaps_size, hipMemcpyDeviceToHost));
//            hipMemset(h_bench.d_bitmaps, 0, bench->bitmaps_size);
//        }

        if(bench->config->bloom_filter){
            BloomFilter_Add<<<bench->config->kv_restriction / 1024 + 1,1024>>>(d_bench);
            check_execution();
            hipDeviceSynchronize();
            CUDA_SAFE_CALL(hipMemcpy(&h_bench, d_bench, sizeof(workbench), hipMemcpyDeviceToHost));
            logt("bloom filter ", start);
            CUDA_SAFE_CALL(hipMemcpy(bench->pstFilter[bench->MemTable_count], h_bench.d_pstFilter, bench->dwFilterSize, hipMemcpyDeviceToHost));
            hipMemset(h_bench.d_pstFilter, 0, bench->dwFilterSize);
        }

        bench->MemTable_count++;

        //init
        int overflow = h_bench.kv_count - bench->config->kv_restriction;
        CUDA_SAFE_CALL(hipMemcpy(h_bench.d_keys, h_bench.d_keys + bench->config->kv_restriction, overflow * sizeof(uint64_t), hipMemcpyDeviceToDevice));              //for the overflow part
        CUDA_SAFE_CALL(hipMemcpy(h_bench.d_values, h_bench.d_values + bench->config->kv_restriction, overflow * sizeof(__uint128_t), hipMemcpyDeviceToDevice));
        h_bench.kv_count = overflow;
        CUDA_SAFE_CALL(hipMemcpy(d_bench, &h_bench, sizeof(workbench), hipMemcpyHostToDevice));                       //update kv_count, other effect ???
        bench->pro.cuda_sort_time += get_time_elapsed(start,false);
        logt("init after sort",start);
    }

    if(bench->crash_consistency){
        cout<<"crash_consistency, 1 cuda sort"<<endl;
        bench->start_time_min = h_bench.start_time_min;
        bench->start_time_max = h_bench.start_time_max;
        h_bench.start_time_min = (1ULL<<32) -1;
        h_bench.start_time_max = 0;
        uint offset = 0;
        if(bench->big_sorted_run_count%2==1){
            offset = bench->config->MemTable_capacity/2;
        }
        if(true){
            write_wid<<<h_bench.kv_count / 1024 + 1,1024>>>(d_bench);
            check_execution();
            hipDeviceSynchronize();
            CUDA_SAFE_CALL(hipMemcpy(&h_bench, d_bench, sizeof(workbench), hipMemcpyDeviceToHost));
            //logt("bloom filter ", start);
            CUDA_SAFE_CALL(hipMemcpy(bench->h_wids[offset+bench->MemTable_count], h_bench.d_wids, bench->config->num_objects*sizeof(unsigned short), hipMemcpyDeviceToHost));      //offset not change

//            hipMemset(h_bench.d_wids, 0, bench->config->num_objects*sizeof(unsigned short));
//            hipMemset(h_bench.same_pid_count, 0, bench->config->num_objects * sizeof(unsigned short));
        }

        // wrap raw pointer with a device_ptr
        thrust::device_ptr<uint64_t> d_vector_keys = thrust::device_pointer_cast(h_bench.d_keys);
        thrust::device_ptr<__uint128_t> d_vector_values = thrust::device_pointer_cast(h_bench.d_values);
        bench->pro.cuda_sort_time += get_time_elapsed(start,false);
        logt("pointer_cast: ",start);
        thrust::sort_by_key(d_vector_keys, d_vector_keys + h_bench.kv_count, d_vector_values);
        check_execution();
        hipDeviceSynchronize();
        CUDA_SAFE_CALL(hipMemcpy(&h_bench, d_bench, sizeof(workbench), hipMemcpyDeviceToHost));
        bench->pro.cuda_sort_time += get_time_elapsed(start,false);
        logt("cuda_sort_time: ",start);
        CUDA_SAFE_CALL(hipMemcpy(bench->h_keys[offset+bench->MemTable_count], h_bench.d_keys, h_bench.kv_count * sizeof(uint64_t), hipMemcpyDeviceToHost));
        CUDA_SAFE_CALL(hipMemcpy(bench->h_values[offset+bench->MemTable_count], h_bench.d_values, h_bench.kv_count * sizeof(__uint128_t), hipMemcpyDeviceToHost));
        bench->pro.cuda_sort_time += get_time_elapsed(start,false);
        logt("hipMemcpy kv",start);
        printf("hipMemcpy kv right\n");

        print_128(bench->h_keys[offset+bench->MemTable_count][10]);
        printf("\n");
        print_128(bench->h_values[offset+bench->MemTable_count][10]);
        printf("\n");
        printf("hipMemcpy kv right\n");
        cout<<"bench->end_time_min:"<<bench->end_time_min<<endl;
        cout<<"wid:"<<(uint)(bench->h_keys[offset+bench->MemTable_count][10] >> 48)<<endl;
        cout<<"pid:"<<(uint)((bench->h_keys[offset+bench->MemTable_count][10] >> 23) & ((1ULL << 25) - 1))<<endl;
        cout<<"end:"<<(uint)((bench->h_keys[offset+bench->MemTable_count][10] >> 8) & ((1ULL << 15) - 1)) + bench->end_time_min<<endl;
        cout<<"count:"<<(uint)(bench->h_keys[offset+bench->MemTable_count][10] & ((1ULL << 8) - 1))<<endl;
        printf("\n");
        cout<<"duration:"<<(uint)(bench->h_values[offset+bench->MemTable_count][10] >> 113)<<endl;
        cout<<"target:"<<(uint)((bench->h_values[offset+bench->MemTable_count][10] >> 88) & ((1ULL << 25) - 1))<<endl;
        box temp_box(bench->h_values[offset+bench->MemTable_count][10]);
        temp_box.print();
    }

	// todo do the data analyzes, for test only, should not copy out so much stuff
	if(bench->config->analyze_grid||bench->config->analyze_reach||bench->config->profile){
		CUDA_SAFE_CALL(hipMemcpy(bench->grid_counter, h_bench.grid_counter,
				bench->grids_stack_capacity*sizeof(uint), hipMemcpyDeviceToHost));
		logt("copy out grid counting data", start);
	}
	if(bench->config->analyze_reach){
		CUDA_SAFE_CALL(hipMemcpy(bench->grids, h_bench.grids,
							bench->grids_stack_capacity*bench->grid_capacity*sizeof(uint), hipMemcpyDeviceToHost));
		CUDA_SAFE_CALL(hipMemcpy(bench->schema, h_bench.schema,
							bench->schema_stack_capacity*sizeof(QTSchema), hipMemcpyDeviceToHost));
		CUDA_SAFE_CALL(hipMemcpy(bench->meeting_buckets, h_bench.meeting_buckets,
							bench->config->num_meeting_buckets*sizeof(meeting_unit), hipMemcpyDeviceToHost));
		bench->schema_stack_index = h_bench.schema_stack_index;
		bench->grids_stack_index = h_bench.grids_stack_index;
		logt("copy out grid, schema, meeting buckets data", start);
	}

	/* 5. update the index */
	if(bench->config->dynamic_schema){
		// update the schema for future processing
		cuda_update_schema_collect<<<bench->schema_stack_capacity/1024+1,1024>>>(d_bench);
		check_execution();
		hipDeviceSynchronize();
		CUDA_SAFE_CALL(hipMemcpy(&h_bench, d_bench, sizeof(workbench), hipMemcpyDeviceToHost));
		if(h_bench.split_list_index>0){
			cuda_update_schema_split<<<h_bench.split_list_index/1024+1,1024>>>(d_bench, h_bench.split_list_index);
			check_execution();
			hipDeviceSynchronize();
		}
		if(h_bench.merge_list_index>0){
			cuda_update_schema_merge<<<h_bench.merge_list_index/1024+1,1024>>>(d_bench, h_bench.merge_list_index);
			check_execution();
			hipDeviceSynchronize();
		}
		CUDA_SAFE_CALL(hipMemcpy(&h_bench, d_bench, sizeof(workbench), hipMemcpyDeviceToHost));
		bench->pro.index_update_time += get_time_elapsed(start,false);
		logt("schema update %d grids", start, h_bench.grids_stack_index);
	}

    /* 6. search kv info */
    if(bench->search_single){
        cuda_search_single_kv<<<h_bench.kv_count/1024+1,1024>>>(d_bench);
        check_execution();
        hipDeviceSynchronize();
        CUDA_SAFE_CALL(hipMemcpy(&h_bench, d_bench, sizeof(workbench), hipMemcpyDeviceToHost));
        bench->single_find_count = h_bench.single_find_count;
        CUDA_SAFE_CALL(hipMemcpy(bench->search_single_list, h_bench.search_single_list, bench->single_find_count*sizeof(search_info_unit), hipMemcpyDeviceToHost));
        bench->pro.cuda_search_single_kv_time += get_time_elapsed(start,false);
        logt("search_single_kv ", start);
    }
    if(bench->search_multi){
        cout<<"before multi_find_count"<<h_bench.multi_find_count<<endl;
        cuda_search_multi_kv<<<h_bench.kv_count/1024+1,1024>>>(d_bench);
        check_execution();
        hipDeviceSynchronize();
        CUDA_SAFE_CALL(hipMemcpy(&h_bench, d_bench, sizeof(workbench), hipMemcpyDeviceToHost));
        bench->multi_find_count = h_bench.multi_find_count;
        cout<<"after multi_find_count"<<h_bench.multi_find_count<<endl;
        CUDA_SAFE_CALL(hipMemcpy(bench->search_multi_list, h_bench.search_multi_list, bench->multi_find_count*sizeof(search_info_unit), hipMemcpyDeviceToHost));
        bench->pro.cuda_search_multi_kv_time += get_time_elapsed(start,false);
        logt("search_multi_kv ", start);
    }

	/* 6. post-process, copy out data*/
//	if(h_bench.meeting_counter>0){
//		bench->meeting_counter = h_bench.meeting_counter;
//		CUDA_SAFE_CALL(hipMemcpy(bench->meetings, h_bench.meetings, min(bench->meeting_capacity, h_bench.meeting_counter)*sizeof(meeting_unit), hipMemcpyDeviceToHost));
//		bench->pro.copy_time += get_time_elapsed(start,false);
//		logt("copy out %d meeting data", start,h_bench.meeting_counter);
//	}
	// clean the device bench for next round of checking
	cuda_cleargrids<<<bench->grids_stack_capacity/1024+1,1024>>>(d_bench);
	cuda_reset_bench<<<1,1>>>(d_bench);
}
