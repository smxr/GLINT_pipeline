#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include "mygpu.h"
#include "cuda_util.h"
#include "../geometry/geometry.h"
#include "../util/query_context.h"
#include "../tracing/partitioner.h"
#include "../tracing/workbench.h"

//
__global__
void partition_cuda(workbench *bench){
	int pid = blockIdx.x*blockDim.x+threadIdx.x;
	if(pid>=bench->config->num_objects){
		return;
	}

	// search the tree to get in which grid
	uint curnode = 0;
	uint gid = 0;

	Point *p = bench->points+pid;
	while(true){
		int loc = (p->y>bench->schema[curnode].mid_y)*2
								+(p->x>bench->schema[curnode].mid_x);
		// is leaf
		if((bench->schema[curnode].children[loc]&1)){
			gid = bench->schema[curnode].children[loc]>>1;
			break;
		}else{
			curnode = bench->schema[curnode].children[loc]>>1;
		}
	}
	uint *cur_grid = bench->grids+(bench->config->grid_capacity+1)*gid;

	// insert current pid to proper memory space of the target gid
	uint cur_loc = atomicAdd(cur_grid,1);
	assert(cur_loc<bench->config->grid_capacity);
	*(cur_grid+1+cur_loc) = pid;
}

__device__
inline void lookup(workbench *bench, uint pid, uint curnode){

	Point *p = bench->points+pid;

	bool top = (p->y>bench->schema[curnode].mid_y-bench->config->y_buffer);
	bool bottom = (p->y<=bench->schema[curnode].mid_y+bench->config->y_buffer);
	bool left = (p->x<=bench->schema[curnode].mid_x+bench->config->x_buffer);
	bool right = (p->x>bench->schema[curnode].mid_x-bench->config->x_buffer);
	uint need_check = (bottom&&left)*1+(bottom&&right)*2+(top&&left)*4+(top&&right)*8;
	for(int i=0;i<4;i++){
		if((need_check>>i)&1){
			if((bench->schema[curnode].children[i]&1)){
				uint gid = bench->schema[curnode].children[i]>>1;
				assert(gid<bench->num_grids);
				uint offset = 0;
				while(offset<bench->grids[gid*(bench->config->grid_capacity+1)]){
					uint cu_index = atomicAdd(&bench->num_checking_units, 1);
					bench->checking_units[cu_index].pid = pid;
					bench->checking_units[cu_index].gid = gid;
					bench->checking_units[cu_index].offset = offset;
					//printf("%d\t%d\t%d\n",pid,gid,offset);
					offset += bench->config->zone_capacity;
				}
			}else{
				lookup(bench, pid, bench->schema[curnode].children[i]>>1);
			}
		}
	}
}

// with recursive call
__global__
void lookup_recursive_cuda(workbench *bench){
	int pid = blockIdx.x*blockDim.x+threadIdx.x;
	if(pid>=bench->config->num_objects){
		return;
	}
	lookup(bench,pid,0);
	return;
}

__global__
void initstack_cuda(workbench *bench){
	int pid = blockIdx.x*blockDim.x+threadIdx.x;
	if(pid>=bench->config->num_objects){
		return;
	}
	uint stack_index = atomicAdd(&bench->stack_index[0],1);
	assert(stack_index<bench->stack_capacity);
	bench->lookup_stack[0][stack_index*2] = pid;
	bench->lookup_stack[0][stack_index*2+1] = 0;
	printf("%d %d %d\n",stack_index,bench->lookup_stack[0][stack_index*2],bench->lookup_stack[0][stack_index*2+1]);
}

__global__
void lookup_cuda(workbench *bench, uint stack_id){

	int sid = blockIdx.x*blockDim.x+threadIdx.x;
	if(sid>=bench->stack_index[stack_id]){
		return;
	}

	uint pid = bench->lookup_stack[stack_id][sid*2];
	uint curnode = bench->lookup_stack[stack_id][sid*2+1];
	Point *p = bench->points+pid;

	// could be possibly in multiple children with buffers enabled
	bool top = (p->y>bench->schema[curnode].mid_y-bench->config->y_buffer);
	bool bottom = (p->y<=bench->schema[curnode].mid_y+bench->config->y_buffer);
	bool left = (p->x<=bench->schema[curnode].mid_x+bench->config->x_buffer);
	bool right = (p->x>bench->schema[curnode].mid_x-bench->config->x_buffer);
	uint need_check = (bottom&&left)*1+(bottom&&right)*2+(top&&left)*4+(top&&right)*8;
	for(int i=0;i<4;i++){
		if((need_check>>i)&1){
			if((bench->schema[curnode].children[i]&1)){
				uint gid = bench->schema[curnode].children[i]>>1;
				assert(gid<bench->num_grids);
				uint offset = 0;
				while(offset<bench->grids[gid*(bench->config->grid_capacity+1)]){
					uint cu_index = atomicAdd(&bench->num_checking_units, 1);
					assert(cu_index<bench->checking_units_capacity);
					bench->checking_units[cu_index].pid = pid;
					bench->checking_units[cu_index].gid = gid;
					bench->checking_units[cu_index].offset = offset;
					//printf("%d\t%d\t%d\n",pid,gid,offset);
					offset += bench->config->zone_capacity;
				}
			}else{
				uint stack_index = atomicAdd(&bench->stack_index[!stack_id],1);
				assert(stack_index<bench->stack_capacity);
				bench->lookup_stack[!stack_id][stack_index*2] = pid;
				bench->lookup_stack[!stack_id][stack_index*2+1] = bench->schema[curnode].children[i]>>1;
			}
		}
	}
	// reset the index to 0
	if(sid == 0){
		bench->stack_index[stack_id] = 0;
	}
}


__global__
void reachability_cuda(workbench *bench){

	// the objects in which grid need be processed
	int pairid = blockIdx.x*blockDim.x+threadIdx.x;
	if(pairid>=bench->num_checking_units){
		return;
	}

	double max_dist = bench->config->reach_distance;
	uint pid = bench->checking_units[pairid].pid;
	uint gid = bench->checking_units[pairid].gid;
	uint offset = bench->checking_units[pairid].offset;
	uint size = *(bench->grids+(bench->config->grid_capacity+1)*gid)-offset;

	if(size>bench->config->zone_capacity){
		size = bench->config->zone_capacity;
	}
	//printf("%d\t%d\t%d\t%d\n",pid,gid,offset,size);

	const uint *cur_pids = bench->grids+(bench->config->grid_capacity+1)*gid+1+offset;
	for(uint i=0;i<size;i++){
		if(pid!=cur_pids[i]){
			double dist = distance(bench->points[pid].x, bench->points[pid].y, bench->points[cur_pids[i]].x, bench->points[cur_pids[i]].y);
			if(dist<=max_dist){
				uint loc = atomicAdd(&bench->num_meeting, (uint)1);
				assert(loc<bench->meeting_capacity);
				bench->meetings[loc].pid1 = pid;
				bench->meetings[loc].pid2 = cur_pids[i];
			}
		}
	}
}


/*
 *
 * check the reachability of objects in a list of partitions
 * ctx.data contains the list of
 *
 * */
void process_with_gpu(workbench *bench){
	struct timeval start = get_cur_time();
	vector<gpu_info *> gpus = get_gpus();
	gpu_info *gpu = gpus[0];
	//gpu->print();
	assert(gpu);

	pthread_mutex_lock(&gpu->lock);
	hipSetDevice(gpu->device_id);

	// use h_bench as a container to copy in and out GPU
	workbench *h_bench = new workbench(bench->config);
	h_bench->num_grids = bench->num_grids;

	// space for the raw points data
	h_bench->points = (Point *)gpu->get_data(0, bench->config->num_objects*sizeof(Point));
	// space for the pids of all the grids
	h_bench->grids = (uint *)gpu->get_data(1, bench->num_grids*(bench->config->grid_capacity+1)*sizeof(uint));
	// space for the pid-zid pairs
	h_bench->checking_units = (checking_unit *)gpu->get_data(2, bench->checking_units_capacity*sizeof(checking_unit));
	// space for the QTtree schema
	h_bench->schema = (QTSchema *)gpu->get_data(3, bench->num_nodes*sizeof(QTSchema));
	// space for processing stack
	h_bench->lookup_stack[0] = (uint *)gpu->get_data(4, bench->stack_capacity*2*sizeof(uint));
	h_bench->lookup_stack[1] = (uint *)gpu->get_data(5, bench->stack_capacity*2*sizeof(uint));
	h_bench->meetings = (meeting_unit *)gpu->get_data(6, bench->meeting_capacity*sizeof(meeting_unit));

	// space for the mapping of bench in GPU
	workbench *d_bench = (workbench *)gpu->get_data(7, sizeof(workbench));
	logt("allocating space %d MB", start,gpu->size_allocated()/1024/1024);

	struct timeval start_execute = get_cur_time();

	CUDA_SAFE_CALL(hipMemcpy(h_bench->points, bench->points, bench->config->num_objects*sizeof(Point), hipMemcpyHostToDevice));
	CUDA_SAFE_CALL(hipMemcpy(h_bench->schema, bench->schema, bench->num_nodes*sizeof(QTSchema), hipMemcpyHostToDevice));
	CUDA_SAFE_CALL(hipMemcpy(d_bench, h_bench, sizeof(workbench), hipMemcpyHostToDevice));
	logt("copying data", start);

	partition_cuda<<<bench->config->num_objects/1024+1,1024>>>(d_bench);
	check_execution();
	hipDeviceSynchronize();
	logt("partition data", start);

	initstack_cuda<<<bench->config->num_objects/1024+1,1024>>>(d_bench);
	check_execution();
	hipDeviceSynchronize();
	CUDA_SAFE_CALL(hipMemcpy(h_bench, d_bench, sizeof(workbench), hipMemcpyDeviceToHost));
	uint stack_id = 0;
	while(h_bench->stack_index[stack_id]>0){
		struct timeval tt = get_cur_time();
		lookup_cuda<<<h_bench->stack_index[stack_id]/1024+1,1024>>>(d_bench,stack_id);
		check_execution();
		hipDeviceSynchronize();
		CUDA_SAFE_CALL(hipMemcpy(h_bench, d_bench, sizeof(workbench), hipMemcpyDeviceToHost));
		stack_id = !stack_id;
	}
	logt("lookup", start);

	// compute the reachability of objects in each partitions
	reachability_cuda<<<h_bench->num_checking_units/1024+1,1024>>>(d_bench);
	check_execution();
	hipDeviceSynchronize();
	logt("computing reachability", start);
	logt("one round",start_execute);

	bench->num_checking_units = h_bench->num_checking_units;
	CUDA_SAFE_CALL(hipMemcpy(bench->grids, h_bench->grids, bench->num_grids*(bench->config->grid_capacity+1)*sizeof(uint), hipMemcpyDeviceToHost));
	CUDA_SAFE_CALL(hipMemcpy(bench->checking_units, h_bench->checking_units, h_bench->num_checking_units*sizeof(checking_unit), hipMemcpyDeviceToHost));
	CUDA_SAFE_CALL(hipMemcpy(bench->meetings, h_bench->meetings, h_bench->num_meeting*sizeof(meeting_unit), hipMemcpyDeviceToHost));


	h_bench->grids = NULL;
	h_bench->checking_units = NULL;
	h_bench->schema = NULL;
	h_bench->lookup_stack[0] = NULL;
	h_bench->lookup_stack[1] = NULL;
	h_bench->meetings = NULL;
	delete h_bench;
	pthread_mutex_unlock(&gpu->lock);
	for(gpu_info *g:gpus){
		delete g;
	}
}
