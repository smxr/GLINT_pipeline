#include "hip/hip_runtime.h"
/*
 * test.cu
 *
 *  Created on: Feb 16, 2021
 *      Author: teng
 */

#include "cuda_util.h"
#include "../geometry/geometry.h"



/*
 * for test
 * */

__global__ void mykernel(Point *p1, Point *p2, double *dist) {
	*dist = distance(p1->x,p1->y,p2->x,p2->y);
	printf("gpu %f\n",*dist);
  //*addr += 10;
  //atomicAdd(addr, 10);       // only available on devices with compute capability 6.x
}

double foo(Point *p1, Point *p2) {
	Point *d_p1,*d_p2;
	double *d_dist;
	hipMallocManaged(&d_p1, sizeof(Point));
	hipMallocManaged(&d_p2, sizeof(Point));
	hipMallocManaged(&d_dist, sizeof(double));

	CUDA_SAFE_CALL(hipMemcpy(d_p1, p1, sizeof(Point), hipMemcpyHostToDevice));
	CUDA_SAFE_CALL(hipMemcpy(d_p2, p2, sizeof(Point), hipMemcpyHostToDevice));

	mykernel<<<1,1>>>(d_p1, d_p2, d_dist);
	double dist = 0;
	CUDA_SAFE_CALL(hipMemcpy(&dist, d_dist, sizeof(double), hipMemcpyDeviceToHost));
	return dist;
}

